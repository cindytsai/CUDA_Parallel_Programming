// Vector addition: C = 1/A + 1/B 
// using multiple GPUs with OpenMP

// Includes
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>          // header for OpenMP
#include <hip/hip_runtime.h>

// Variables
float* h_A;   // host vectors
float* h_B;
float* h_C;
float* h_D;

// Functions
void RandomInit(float*, int);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = 1.0/A[i] + 1.0/B[i];
    __syncthreads();
}

// Host code

int main(void)
{
    printf("\n");
    printf("Vector Addition with multiple GPUs \n");
    int N, NGPU, cpu_thread_id=0;
    int *Dev; 
    long mem = 1024*1024*1024;     // 4 Giga for float data type.

    printf("Enter the number of GPUs: ");
    scanf("%d", &NGPU);
    printf("%d\n", NGPU);
    Dev = (int *)malloc(sizeof(int)*NGPU);

    int numDev = 0;
    printf("GPU device number: ");
    for(int i = 0; i < NGPU; i++) {
      scanf("%d", &Dev[i]);
      printf("%d ",Dev[i]);
      numDev++;
      if(getchar() == '\n') break;
    }
    printf("\n");
    if(numDev != NGPU) {
      fprintf(stderr,"Should input %d GPU device numbers\n", NGPU);
      exit(1);
    }

    printf("Enter the size of the vectors: ");
    scanf("%d", &N);        
    printf("%d\n", N);        
    if (3*N > mem) {
        printf("The size of these 3 vectors cannot be fitted into 4 Gbyte\n");
        exit(1);
    }
    long size = N*sizeof(float);

    // Set the sizes of threads and blocks
    int threadsPerBlock;
    printf("Enter the number of threads per block: ");
    scanf("%d", &threadsPerBlock);
    printf("%d\n", threadsPerBlock);
    if(threadsPerBlock > 1024) {
      printf("The number of threads per block must be less than 1024 ! \n");
      exit(1);
    }
    int blocksPerGrid = (N + threadsPerBlock*NGPU - 1) / (threadsPerBlock*NGPU);
    printf("The number of blocks is %d\n", blocksPerGrid);
    if(blocksPerGrid > 2147483647) {
      printf("The number of blocks must be less than 2147483647 ! \n");
      exit(1);
    }

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    if (! h_A || ! h_B || ! h_C) {
	printf("!!! Not enough memory.\n");
	exit(1);
    }
    
    // Initialize input vectors

    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // declare cuda event for timer
    hipEvent_t start, stop;
//    cudaEventCreate(&start);    // events must be created after devices are set 
//    cudaEventCreate(&stop);

    float Intime,gputime,Outime;

    // Set numbers of threads = numbers of GPU
    omp_set_num_threads(NGPU);

    // So that "cpu_thread_id" is declared under each threads, and they are independent.
    #pragma omp parallel private(cpu_thread_id)
    {
    	float *d_A, *d_B, *d_C;
    	cpu_thread_id = omp_get_thread_num();
    	hipSetDevice(Dev[cpu_thread_id]);


        // start the timer
        // And maybe since OpenMP thread id = 0 , start the first (?)
        // Start the clock here, to see how much time it takes to input array.
        // And also, we use a thread (here '0') to track the clock.
        if(cpu_thread_id == 0) {
          hipEventCreate(&start);
          hipEventCreate(&stop);
          hipEventRecord(start,0);
        }

    	// Allocate vectors in device memory
        // size should be devisiable by Number of GPU 
    	hipMalloc((void**)&d_A, size/NGPU);
    	hipMalloc((void**)&d_B, size/NGPU);
    	hipMalloc((void**)&d_C, size/NGPU);

        // Copy vectors from host memory to device memory
    	hipMemcpy(d_A, h_A+N/NGPU*cpu_thread_id, size/NGPU, hipMemcpyHostToDevice);
    	hipMemcpy(d_B, h_B+N/NGPU*cpu_thread_id, size/NGPU, hipMemcpyHostToDevice);

        // Wait until all threads come to this step, synchronizes all threads on OpenMP
    	#pragma omp barrier

        // stop the timer
    	if(cpu_thread_id == 0) {
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);
            hipEventElapsedTime( &Intime, start, stop);
            printf("Data input time for GPU: %f (ms) \n",Intime);
    	}

        // start the timer
        if(cpu_thread_id == 0) hipEventRecord(start,0);

        VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N/NGPU);
        // Blocks until the device has completed all the preceding requested task.
    	hipDeviceSynchronize();

        // stop the timer
    	if(cpu_thread_id == 0) {
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);
            hipEventElapsedTime( &gputime, start, stop);
            printf("Processing time for GPU: %f (ms) \n",gputime);
            printf("GPU Gflops: %f\n",3*N/(1000000.0*gputime));
    	}

        // Copy result from device memory to host memory
        // h_C contains the result in host memory
        // start the timer
        if(cpu_thread_id == 0) hipEventRecord(start,0);
        hipMemcpy(h_C+N/NGPU*cpu_thread_id, d_C, size/NGPU, hipMemcpyDeviceToHost);
    	hipFree(d_A);
    	hipFree(d_B);
    	hipFree(d_C);

        // stop the timer
    	if(cpu_thread_id == 0) {
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);
            hipEventElapsedTime( &Outime, start, stop);
            printf("Data output time for GPU: %f (ms) \n",Outime);
    	}
    }

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n",gputime_tot);

    // start the timer
    hipEventRecord(start,0);

    h_D = (float*)malloc(size);     // compute the reference solution
    for (int i = 0; i < N; ++i) {
        h_D[i] = 1.0/h_A[i] + 1.0/h_B[i];
    }
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",3*N/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/gputime_tot);

    // Destroy timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result
    printf("Check result:\n");
    double sum=0; 
    double diff;
    for (int i = 0; i < N; ++i) {
        diff = abs(h_D[i] - h_C[i]);
        sum += diff*diff; 
    }
    sum = sqrt(sum);
    printf("norm(h_C - h_D)=%20.15e\n",sum);

    for (int i=0; i < NGPU; i++) {
	hipSetDevice(i);
	hipDeviceReset();
    }

    return 0;
}


// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
