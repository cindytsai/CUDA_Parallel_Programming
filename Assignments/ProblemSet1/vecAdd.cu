#include "hip/hip_runtime.h"
// Vector addition: C = 1/A + 1/B.
// compile with the following command:
//
// (for GTX970)
// nvcc -arch=compute_52 -code=sm_52,sm_52 -O2 -m64 -o vecAdd vecAdd.cu
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O2 -m64 -o vecAdd vecAdd.cu


// Includes
#include <stdio.h>
#include <stdlib.h>

// Variables
float* h_A;   // host vectors
float* h_B;
float* h_C;
float* h_D;
float* d_A;   // device vectors
float* d_B;
float* d_C;

// Functions
void RandomInit(float*, int);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = 1.0/A[i] + 1.0/B[i];
    
    __syncthreads();

}

// Host code

int main( )
{

    int gid;   

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    scanf("%d",&gid);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Set GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("Vector Addition: C = 1/A + 1/B\n");
    int mem = 1024*1024*1024;     // Giga    
    int N;

    printf("Enter the size of the vectors: ");
    scanf("%d",&N);        
    printf("%d\n",N);        
    if( 3*N > mem ) {     // each real number takes 4 bytes
      printf("The size of these 3 vectors cannot be fitted into 4 Gbyte\n");
      exit(2);
    }
    long size = N * sizeof(float);


    // Allocate input vectors h_A and h_B in host memory

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // Initialize the input vectors with random numbers

    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // Set the sizes of threads and blocks

    int threadsPerBlock;
loop:
    printf("Enter the number of threads per block: ");
    scanf("%d",&threadsPerBlock);
    printf("%d\n",threadsPerBlock);
    if( threadsPerBlock > 1024 ) {
      printf("The number of threads per block must be less than 1024 ! \n");
      goto loop;
    }
    int blocksPerGrid = (N + threadsPerBlock - 1)/threadsPerBlock;
    printf("The number of blocks is %d\n", blocksPerGrid);
    if( blocksPerGrid > 2147483647 ) {
      printf("The number of blocks must be less than 2147483647 ! \n");
      goto loop;
    }

    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start,0);

    // Allocate vectors in device memory

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy vectors from host memory to device memory

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Intime;
    hipEventElapsedTime( &Intime, start, stop);
    printf("Input time for GPU: %f (ms) \n",Intime);

    // start the timer
    hipEventRecord(start,0);

    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float gputime;
    daEventElapsedTime( &gputime, start, stop);cu
    printf("Processing time for GPU: %f (ms) \n",gputime);
    printf("GPU Gflops: %f\n",3*N/(1000000.0*gputime));
    
    // Copy result from device memory to host memory
    // h_C contains the result in host memory

    // start the timer
    hipEventRecord(start,0);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Outime;
    hipEventElapsedTime( &Outime, start, stop);
    printf("Output time for GPU: %f (ms) \n",Outime);

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n",gputime_tot);

    // start the timer
    hipEventRecord(start,0);

    h_D = (float*)malloc(size);       // to compute the reference solution
    for (int i = 0; i < N; ++i) 
        h_D[i] = 1.0/h_A[i] + 1.0/h_B[i];
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",3*N/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/(gputime_tot));

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result

    printf("Check result:\n");
    double sum=0; 
    double diff;
    for (int i = 0; i < N; ++i) {
      diff = abs(h_D[i] - h_C[i]);
      sum += diff*diff; 
//      if(diff > 1.0e-15) { 
//        printf("i=%d, h_D=%15.10e, h_C=%15.10e \n", i, h_D[i], h_C[i]);
//      }
    }
    sum = sqrt(sum);
    printf("norm(h_C - h_D)=%20.15e\n\n",sum);

    hipDeviceReset();
}


// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{   
    for(int i = 0; i< n; i++)
        data[i] = rand() / (float)RAND_MAX;
}



