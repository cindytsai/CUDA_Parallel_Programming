// Solve the Laplace equation on a 2D lattice with boundary conditions.
//
// compile with the following command:
//
// (for GTX970)
// nvcc -arch=compute_52 -code=sm_52,sm_52 -O3 -m64 -o laplace laplace.cu
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O3 -m64 -o laplace laplace.cu


// Includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// field variables
float* h_new;   // host field vectors
float* h_old;   
float* h_C;     // result of diff*diff of each block
float* g_new;   
float* d_new;   // device field vectors
float* d_old;  
float* d_C;

int     MAX=1000000;      // maximum iterations
double  eps=1.0e-10;      // stopping criterion


__global__ void laplacian(float* phi_old, float* phi_new, float* C, bool flag)
{
    extern __shared__ float cache[];     
    float  t, l, r, b;     // top, left, right, bottom
    float  diff; 
    int    site, ym1, xm1, xp1, yp1;

    int Nx = blockDim.x*gridDim.x; // number of site in x direction
    int Ny = blockDim.y*gridDim.y; // number of site in y direction
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    int cacheIndex = threadIdx.x + threadIdx.y*blockDim.x;  

    site = x + y*Nx;

    if((x == 0) || (x == Nx-1) || (y == 0) || (y == Ny-1) ) {  
    }
    else {
      xm1 = site - 1;    // x-1
      xp1 = site + 1;    // x+1
      ym1 = site - Nx;   // y-1
      yp1 = site + Nx;   // y+1
      if(flag) {
        b = phi_old[ym1]; 
        l = phi_old[xm1]; 
        r = phi_old[xp1]; 
        t = phi_old[yp1]; 
        phi_new[site] = 0.25*(b+l+r+t);
      }
      else {
        b = phi_new[ym1]; 
        l = phi_new[xm1]; 
        r = phi_new[xp1]; 
        t = phi_new[yp1]; 
        phi_old[site] = 0.25*(b+l+r+t);
      }
      diff = phi_new[site]-phi_old[site];
    }
    cache[cacheIndex]=diff*diff;
    __syncthreads();

    // perform parallel reduction

    int ib = blockDim.x*blockDim.y/2;  
    while (ib != 0) {  
      if(cacheIndex < ib)  
        cache[cacheIndex] += cache[cacheIndex + ib];
      __syncthreads();
      ib /=2;  
    } 
    int blockIndex = blockIdx.x + gridDim.x*blockIdx.y;
    if(cacheIndex == 0)  C[blockIndex] = cache[0];
}

int main(void)
{

    int gid;              // GPU_ID
    int iter;
    volatile bool flag;   // to toggle between *_new and *_old  
    float cputime;
    float gputime;
    float gputime_tot;
    double flops;
    double error;
    
    printf("Enter the GPU ID (0/1): ");
    scanf("%d",&gid);
    printf("%d\n",gid);

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Select GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("Solve Laplace equation on a 2D lattice with boundary conditions\n");

    int Nx,Ny;                // lattice size
    printf("Enter the size (Nx, Ny) of the 2D lattice: ");
    scanf("%d %d",&Nx,&Ny);        
    printf("%d %d\n",Nx,Ny);        

    // Set the number of threads (tx,ty) per block
   
    int tx,ty;
    printf("Enter the number of threads (tx,ty) per block: ");
    scanf("%d %d",&tx, &ty);
    printf("%d %d\n",tx, ty);
    if( tx*ty > 1024 ) {
      printf("The number of threads per block must be less than 1024 ! \n");
      exit(0);
    }
    dim3 threads(tx,ty); 
    
    // The total number of threads in the grid is equal to the total number of lattice sites
    
    int bx = Nx/tx;
    if(bx*tx != Nx) {
      printf("The block size in x is incorrect\n"); 
      exit(0);
    }
    int by = Ny/ty;
    if(by*ty != Ny) {
      printf("The block size in y is incorrect\n"); 
      exit(0);
    }
    if((bx > 65535)||(by > 65535)) {
      printf("The grid size exceeds the limit ! \n");
      exit(0);
    }
    dim3 blocks(bx,by);
    printf("The dimension of the grid is (%d, %d)\n",bx,by); 

    int CPU;    
    printf("To compute the solution vector with CPU/GPU/both (0/1/2) ? ");
    scanf("%d",&CPU);
    printf("%d\n",CPU);
    fflush(stdout);

    // Allocate field vector h_phi in host memory

    int N = Nx*Ny;
    int size = N*sizeof(float);
    int sb = bx*by*sizeof(float);
    h_old = (float*)malloc(size);
    h_new = (float*)malloc(size);
    g_new = (float*)malloc(size);
    h_C = (float*)malloc(sb);
   
    // Initialize the array to 0
    memset(h_old, 0, size);
    memset(h_new, 0, size);

    // Initialize the field vector with boundary conditions
    for(int x=0; x<Nx; x++) {
      h_new[x+Nx*(Ny-1)]=1.0;  
      h_old[x+Nx*(Ny-1)]=1.0;
    }  

    FILE *out1;                 // save initial configuration in phi_initial.dat
    out1 = fopen("phi_initial.dat","w");

    fprintf(out1, "Inital field configuration:\n");
    for(int j=Ny-1;j>-1;j--) {
      for(int i=0; i<Nx; i++) {
        fprintf(out1,"%.2e ",h_new[i+j*Nx]);
      }
      fprintf(out1,"\n");
    }
    fclose(out1);

//    printf("\n");
//    printf("Inital field configuration:\n");
//    for(int j=Ny-1;j>-1;j--) {
//      for(int i=0; i<Nx; i++) {
//        printf("%.2e ",h_new[i+j*Nx]);
//      }
//      printf("\n");
//    }

    printf("\n");

    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if(CPU>0) {

      // start the timer
      hipEventRecord(start,0);

      // Allocate vectors in device memory

      hipMalloc((void**)&d_new, size);
      hipMalloc((void**)&d_old, size);
      hipMalloc((void**)&d_C, sb);
  
      // Copy vectors from host memory to device memory

      hipMemcpy(d_new, h_new, size, hipMemcpyHostToDevice);
      hipMemcpy(d_old, h_old, size, hipMemcpyHostToDevice);
    
      // stop the timer
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      float Intime;
      hipEventElapsedTime( &Intime, start, stop);
      printf("Input time for GPU: %f (ms) \n",Intime);

      // start the timer
      hipEventRecord(start,0);

      error = 10*eps;  // any value bigger than eps is OK
      iter = 0;        // counter for iterations
      flag = true; 
 
      int sm = tx*ty*sizeof(float);   // size of the shared memory in each block

      while ( (error > eps) && (iter < MAX) ) {

        laplacian<<<blocks,threads,sm>>>(d_old, d_new, d_C, flag);
        hipMemcpy(h_C, d_C, sb, hipMemcpyDeviceToHost);
        error = 0.0;
        for(int i=0; i<bx*by; i++) {
          error = error + h_C[i];
        }
        error = sqrt(error);

//        printf("error = %.15e\n",error);
//        printf("iteration = %d\n",iter);

        iter++;
        flag = !flag;

      }

      printf("error (GPU) = %.15e\n",error);
      printf("total iterations (GPU) = %d\n",iter);
    
      // stop the timer
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      hipEventElapsedTime( &gputime, start, stop);
      printf("Processing time for GPU: %f (ms) \n",gputime);
      flops = 7.0*(Nx-2)*(Ny-2)*iter;
      printf("GPU Gflops: %f\n",flops/(1000000.0*gputime));
    
      // Copy result from device memory to host memory
  
      // start the timer
      hipEventRecord(start,0);

      // Because after the iteration, d_new and d_old are basically the same.
      hipMemcpy(g_new, d_new, size, hipMemcpyDeviceToHost);

      hipFree(d_new);
      hipFree(d_old);
      hipFree(d_C);

      // stop the timer
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      float Outime;
      hipEventElapsedTime( &Outime, start, stop);
      printf("Output time for GPU: %f (ms) \n",Outime);

      gputime_tot = Intime + gputime + Outime;
      printf("Total time for GPU: %f (ms) \n",gputime_tot);
      fflush(stdout);

      FILE *outg;                 // save GPU solution in phi_GPU.dat
      outg = fopen("phi_GPU.dat","w");

      fprintf(outg, "GPU field configuration:\n");
      for(int j=Ny-1;j>-1;j--) {
        for(int i=0; i<Nx; i++) {
          fprintf(outg,"%.2e ",g_new[i+j*Nx]);
        }
        fprintf(outg,"\n");
      }
      fclose(outg);

//      printf("\n");
//      printf("Final field configuration (GPU):\n");
//      for(int j=Ny-1;j>-1;j--) {
//        for(int i=0; i<Nx; i++) {
//          printf("%.2e ",g_new[i+j*Nx]);
//        }
//        printf("\n");
//      }

      printf("\n");

    } 

    if(CPU==1) {      // not to compute the CPU solution 
      free(h_new);
      free(h_old);
      free(g_new);
      free(h_C);
      hipDeviceReset();
      exit(0);
    }
 
    if((CPU==0)||(CPU==2)) {      // to compute the CPU solution 

      // start the timer
      hipEventRecord(start,0);

      // to compute the reference solution

      error = 10*eps;      // any value bigger than eps 
      iter = 0;            // counter for iterations
      flag = true;     
      double diff; 

      float t, l, r, b;    // top, left, right, bottom
      int site, ym1, xm1, xp1, yp1;

      while ( (error > eps) && (iter < MAX) ) {
        if(flag) {
          error = 0.0;
          for(int y=0; y<Ny; y++) {
          for(int x=0; x<Nx; x++) { 
            if(x==0 || x==Nx-1 || y==0 || y==Ny-1) {   
            }
            else {
              site = x+y*Nx;
              xm1 = site - 1;    // x-1
              xp1 = site + 1;    // x+1
              ym1 = site - Nx;   // y-1
              yp1 = site + Nx;   // y+1
              b = h_old[ym1]; 
              l = h_old[xm1]; 
              r = h_old[xp1]; 
              t = h_old[yp1]; 
              h_new[site] = 0.25*(b+l+r+t);
              diff = h_new[site]-h_old[site]; 
              error = error + diff*diff;
            }
          } 
          } 
        }
        else {
          error = 0.0;
          for(int y=0; y<Ny; y++) {
          for(int x=0; x<Nx; x++) { 
            if(x==0 || x==Nx-1 || y==0 || y==Ny-1) {
            }
            else {
              site = x+y*Nx;
              xm1 = site - 1;    // x-1
              xp1 = site + 1;    // x+1
              ym1 = site - Nx;   // y-1
              yp1 = site + Nx;   // y+1
              b = h_new[ym1]; 
              l = h_new[xm1]; 
              r = h_new[xp1]; 
              t = h_new[yp1]; 
              h_old[site] = 0.25*(b+l+r+t);
              diff = h_new[site]-h_old[site]; 
              error = error + diff*diff;
            } 
          }
          }
        }
        flag = !flag;
        iter++;
        error = sqrt(error);

//        printf("error = %.15e\n",error);
//        printf("iteration = %d\n",iter);

      }                   // exit if error < eps
    
      printf("error (CPU) = %.15e\n",error);
      printf("total iterations (CPU) = %d\n",iter);

      // stop the timer
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      hipEventElapsedTime( &cputime, start, stop);
      printf("Processing time for CPU: %f (ms) \n",cputime);
      flops = 7.0*(Nx-2)*(Ny-2)*iter;
      printf("CPU Gflops: %lf\n",flops/(1000000.0*cputime));
      printf("Speed up of GPU = %f\n", cputime/(gputime_tot));
      fflush(stdout);

      // destroy the timer
      hipEventDestroy(start);
      hipEventDestroy(stop);

      FILE *outc;                 // save CPU solution in phi_CPU.dat
      outc = fopen("phi_CPU.dat","w");

      fprintf(outc, "CPU field configuration:\n");
        for(int j=Ny-1;j>-1;j--) {
        for(int i=0; i<Nx; i++) {
          fprintf(outc,"%.2e ",h_new[i+j*Nx]);
        }
        fprintf(outc,"\n");
      }
      fclose(outc);

 //     printf("\n");
 //     printf("Final field configuration (CPU):\n");
 //     for(int j=Ny-1;j>-1;j--) {
 //       for(int i=0; i<Nx; i++) {
 //         printf("%.2e ",h_new[i+j*Nx]);
 //       }
 //       printf("\n");
 //     }

      printf("\n");

      free(h_new);
      free(h_old);
      free(g_new);
      free(h_C);

    }

    hipDeviceReset();
    
}

