// Solve the Laplace equation on a 2D lattice with boundary conditions.
//
// compile with the following command:
//
// (for GTX970)
// nvcc -arch=compute_52 -code=sm_52,sm_52 -O3 -m64 -o laplace laplace.cu
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O3 -m64 -o laplace laplace.cu


// Includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// field variables
float* h_new;   // host field vectors
float* h_old;   
float* h_C;     // result of diff*diff of each block
float* g_new;   
float* d_new;   // device field vectors
float* d_old;  
float* d_C;

int     MAX=1000000;      // maximum iterations
double  eps=1.0e-10;      // stopping criterion


__global__ void laplacian(float* phi_old, float* phi_new, float* C, bool flag)
{
    extern __shared__ float cache[];     
    float  t, l, r, b;     // top, left, right, bottom
    float  diff; 
    int    site, ym1, xm1, xp1, yp1;

    int Nx = blockDim.x*gridDim.x; // number of site in x direction
    int Ny = blockDim.y*gridDim.y; // number of site in y direction
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    int cacheIndex = threadIdx.x + threadIdx.y*blockDim.x;  

    site = x + y*Nx;

    if((x == 0) || (x == Nx-1) || (y == 0) || (y == Ny-1) ) {  
    }
    else {
      xm1 = site - 1;    // x-1
      xp1 = site + 1;    // x+1
      ym1 = site - Nx;   // y-1
      yp1 = site + Nx;   // y+1
      if(flag) {
        b = phi_old[ym1]; 
        l = phi_old[xm1]; 
        r = phi_old[xp1]; 
        t = phi_old[yp1]; 
        phi_new[site] = 0.25*(b+l+r+t);
      }
      else {
        b = phi_new[ym1]; 
        l = phi_new[xm1]; 
        r = phi_new[xp1]; 
        t = phi_new[yp1]; 
        phi_old[site] = 0.25*(b+l+r+t);
      }
      diff = phi_new[site]-phi_old[site];
    }
    cache[cacheIndex]=diff*diff;
    __syncthreads();

    // perform parallel reduction

    int ib = blockDim.x*blockDim.y/2;  
    while (ib != 0) {  
      if(cacheIndex < ib)  
        cache[cacheIndex] += cache[cacheIndex + ib];
      __syncthreads();
      ib /=2;  
    } 
    int blockIndex = blockIdx.x + gridDim.x*blockIdx.y;
    if(cacheIndex == 0)  C[blockIndex] = cache[0];
}

int main(void)
{

    int gid;              // GPU_ID
    int iter;
    volatile bool flag;   // to toggle between *_new and *_old  
    float cputime;
    float gputime;
    float gputime_tot;
    double flops;
    double error;
    
    printf("Enter the GPU ID (0/1): ");
    scanf("%d",&gid);
    printf("%d\n",gid);

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Select GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    int Nx, Ny;   // lattice size
    int tx, ty;   // block size, threads (tx, ty) per block
    int bx, by;   // grid size, block (bx, by) per grid
    int N;        // total number of site
    int size;     // size of the array h_old, h_new
    int sb;       // size of the array h_C;
    int sm;       // size of shared memory
    float Intime;
    float Outime;
    double diff;
    float t, l, r, b;    // top, left, right, bottom
    int site, ym1, xm1, xp1, yp1;

    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Write all the result to file.
    FILE *output;
    output = fopen("Output_CPU_GPU.txt", "a");
    fprintf(output, "LatticeSize BlockSize GPUInput GPUerror GPUiteration GPUonly GPUflop GPUoutput GPUtotal ");
    fprintf(output, "CPUerror CPUiteration CPUonly CPUflop SpeedUp\n");
    fclose(output);

    for(int m = 5; m <= 8; m = m+1){
      for(int n = 2; n <= 5; n = n+1){
        Nx = pow(2, m);
        Ny = Nx;
        tx = pow(2, n);
        ty = tx;

        dim3 threads(tx,ty);
        bx = Nx / tx;
        by = Ny / ty;
        dim3 blocks(bx,by);

        // Allocate field vector h_phi in host memory
        N = Nx * Ny;
        size = N * sizeof(float);
        sb = bx * by * sizeof(float);
        h_old = (float*)malloc(size);
        h_new = (float*)malloc(size);
        g_new = (float*)malloc(size);
        h_C = (float*)malloc(sb);

        // Initialize the array to 0
        memset(h_old, 0, size);
        memset(h_new, 0, size);

        // Initialize the field vector with boundary conditions
        for(int x=0; x<Nx; x++) {
          h_new[x+Nx*(Ny-1)]=1.0;  
          h_old[x+Nx*(Ny-1)]=1.0;
        }

        FILE *out1;                 // save initial configuration in phi_initial.dat
        out1 = fopen("phi_initial.dat","w");

        fprintf(out1, "Inital field configuration:\n");
        for(int j=Ny-1;j>-1;j--) {
          for(int i=0; i<Nx; i++) {
            fprintf(out1,"%.2e ",h_new[i+j*Nx]);
          }
          fprintf(out1,"\n");
        }
        fclose(out1);

        printf("\n");

        // start the timer
        hipEventRecord(start,0);

        // Allocate vectors in device memory

        hipMalloc((void**)&d_new, size);
        hipMalloc((void**)&d_old, size);
        hipMalloc((void**)&d_C, sb);
    
        // Copy vectors from host memory to device memory

        hipMemcpy(d_new, h_new, size, hipMemcpyHostToDevice);
        hipMemcpy(d_old, h_old, size, hipMemcpyHostToDevice);
      
        // stop the timer
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);

        
        hipEventElapsedTime( &Intime, start, stop);
        printf("Input time for GPU: %f (ms) \n",Intime);

        // start the timer
        hipEventRecord(start,0);

        error = 10*eps;  // any value bigger than eps is OK
        iter = 0;        // counter for iterations
        flag = true; 
   
        sm = tx * ty * sizeof(float);   // size of the shared memory in each block

        while ( (error > eps) && (iter < MAX) ) {

          laplacian<<<blocks,threads,sm>>>(d_old, d_new, d_C, flag);
          hipMemcpy(h_C, d_C, sb, hipMemcpyDeviceToHost);
          error = 0.0;
          for(int i=0; i<bx*by; i++) {
            error = error + h_C[i];
          }
          error = sqrt(error);

  //        printf("error = %.15e\n",error);
  //        printf("iteration = %d\n",iter);

          iter++;
          flag = !flag;

        }

        printf("error (GPU) = %.15e\n",error);
        printf("total iterations (GPU) = %d\n",iter);
      
        // stop the timer
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);

        hipEventElapsedTime( &gputime, start, stop);
        printf("Processing time for GPU: %f (ms) \n",gputime);
        flops = 7.0*(Nx-2)*(Ny-2)*iter;
        printf("GPU Gflops: %f\n",flops/(1000000.0*gputime));
      
        // Copy result from device memory to host memory
    
        // start the timer
        hipEventRecord(start,0);

        // Because after the iteration, d_new and d_old are basically the same.
        hipMemcpy(g_new, d_new, size, hipMemcpyDeviceToHost);

        hipFree(d_new);
        hipFree(d_old);
        hipFree(d_C);

        // stop the timer
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);

        
        hipEventElapsedTime( &Outime, start, stop);
        printf("Output time for GPU: %f (ms) \n",Outime);

        gputime_tot = Intime + gputime + Outime;
        printf("Total time for GPU: %f (ms) \n",gputime_tot);
        fflush(stdout);

        FILE *outg;                 // save GPU solution in phi_GPU.dat
        outg = fopen("phi_GPU.dat","w");

        fprintf(outg, "GPU field configuration:\n");
        for(int j=Ny-1;j>-1;j--) {
          for(int i=0; i<Nx; i++) {
            fprintf(outg,"%.2e ",g_new[i+j*Nx]);
          }
          fprintf(outg,"\n");
        }
        fclose(outg);

        // Write all the output to file
        output = fopen("Output_CPU_GPU.txt", "a");        
        fprintf(output, "%d %d %f %f %d %f %f %f %f ", Nx, tx, Intime, error, iter, gputime, flops/(1000000.0*gputime), Outime, gputime_tot);
        fclose(output);

        printf("\n");

        // Compute with CPU only
        // 
        // start the timer
        hipEventRecord(start,0);
        // to compute the reference solution
        error = 10*eps;      // any value bigger than eps 
        iter = 0;            // counter for iterations
        flag = true;

        while ( (error > eps) && (iter < MAX) ) {
          if(flag) {
            error = 0.0;
            for(int y=0; y<Ny; y++) {
              for(int x=0; x<Nx; x++) { 
                if(x==0 || x==Nx-1 || y==0 || y==Ny-1) {   
                }
                else {
                  site = x+y*Nx;
                  xm1 = site - 1;    // x-1
                  xp1 = site + 1;    // x+1
                  ym1 = site - Nx;   // y-1
                  yp1 = site + Nx;   // y+1
                  b = h_old[ym1]; 
                  l = h_old[xm1]; 
                  r = h_old[xp1]; 
                  t = h_old[yp1]; 
                  h_new[site] = 0.25*(b+l+r+t);
                  diff = h_new[site]-h_old[site]; 
                  error = error + diff*diff;
                }
              } 
            } 
          }
          else {
            error = 0.0;
            for(int y=0; y<Ny; y++) {
              for(int x=0; x<Nx; x++) { 
                if(x==0 || x==Nx-1 || y==0 || y==Ny-1) {
                }
                else {
                  site = x+y*Nx;
                  xm1 = site - 1;    // x-1
                  xp1 = site + 1;    // x+1
                  ym1 = site - Nx;   // y-1
                  yp1 = site + Nx;   // y+1
                  b = h_new[ym1]; 
                  l = h_new[xm1]; 
                  r = h_new[xp1]; 
                  t = h_new[yp1]; 
                  h_old[site] = 0.25*(b+l+r+t);
                  diff = h_new[site]-h_old[site]; 
                  error = error + diff*diff;
                } 
              }
            }
          }
          flag = !flag;
          iter++;
          error = sqrt(error);

        }                   // exit if error < eps


        printf("error (CPU) = %.15e\n",error);
        printf("total iterations (CPU) = %d\n",iter);

        // stop the timer
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);

        hipEventElapsedTime( &cputime, start, stop);
        printf("Processing time for CPU: %f (ms) \n",cputime);
        flops = 7.0*(Nx-2)*(Ny-2)*iter;
        printf("CPU Gflops: %lf\n",flops/(1000000.0*cputime));
        printf("Speed up of GPU = %f\n", cputime/(gputime_tot));
        fflush(stdout);

        FILE *outc;                 // save CPU solution in phi_CPU.dat
        outc = fopen("phi_CPU.dat","w");

        fprintf(outc, "CPU field configuration:\n");
        for(int j=Ny-1;j>-1;j--) {
          for(int i=0; i<Nx; i++) {
            fprintf(outc,"%.2e ",h_new[i+j*Nx]);
          }
          fprintf(outc,"\n");
        }
        fclose(outc);

        // Write all the output to file
        output = fopen("Output_CPU_GPU.txt", "a");        
        fprintf(output, "%f %d %f %f %f \n", error, iter, cputime, flops/(1000000.0*cputime), cputime/gputime_tot);
        fclose(output);

        printf("Finish computing lattice size : %d, block size : %d\n", Nx, tx);

        free(h_new);
        free(h_old);
        free(g_new);
        free(h_C);

      }
    }

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipDeviceReset();
    
}

