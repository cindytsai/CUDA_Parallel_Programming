// To compute histogram with atomic operations */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


// Variables
float* data_h;          // host vectors
unsigned int* hist_h;   // GPU solution back to the CPU 
float* data_d;          // device vectors
unsigned int* hist_d;
unsigned int* hist_c;   // CPU solution


// Functions
void RandomUniform(float*, long);
void RandomNormal(float*, long);
void RandomExpDecay(float*, long);

__global__ void hist_shmem(float *data, const long N, unsigned int *hist, 
                           const int bins, const float Rmin, const float binsize) 
{

    // use shared memory and atomic addition

    extern __shared__  unsigned int temp[];     // assume the blocksize is equal to the total # bins
    temp[threadIdx.x] = 0;
    __syncthreads();

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    long stride = blockDim.x * gridDim.x;

//    if( (index > bins-1) || (index < 0)) {
//      printf("data[%d]=%f, index=%d\n",i,data[i],index);
//    }

    while (i < N) {
        int index = (int)((data[i] - Rmin) / binsize);
        atomicAdd(&temp[index], 1);
        i += stride;
    }

    __syncthreads();
    atomicAdd( &(hist[threadIdx.x]), temp[threadIdx.x] );

}


int main(void)
{

    int gid;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    scanf("%d",&gid);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Set GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("To find the histogram of a data set (with real numbers): \n");
    long N;                     // Size of the data vector
    int bins;                   // Number of bins of the histogram
    int index;                  // index for the histogram
    float Rmin, Rmax;           // Range of the distribution of the random number
    float binsize;              // Range of each bins

    printf("Enter the size of the data vector: ");
    scanf("%ld",&N);
    printf("%ld\n",N);
    long size = N * sizeof(float);  // size of the data vector

    printf("Enter the data range [Rmin, Rmax] for the histogram: ");
    scanf("%f %f",&Rmin, &Rmax);
    printf("%f %f\n",Rmin, Rmax);
    fflush(stdout);
    
    //Set Seed for rand() Function.
    srand(12345);

    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threadsPerBlock;
    int blocksPerGrid;
    int sm;

    int bsize;

    float Intime;
    float gputime;
    float Outime;
    float gputime_tot;
    float cputime;

    FILE *output;
    output = fopen("hist_shmem_result.txt", "a");
    fprintf(output, "Bins BlockSize GridSize GPUonly TotalGPU CPUonly SpeedUp\n");
    fclose(output);   

    for(int n = 1; n <= 10; n = n+1){
        for(int m = 1; m <= 6; m = m+1){

            threadsPerBlock = pow(2, n);
            blocksPerGrid = pow(10, m);

            bins = threadsPerBlock;
            bsize = bins * sizeof(int);
            binsize = (Rmax - Rmin) / (float)bins;

            data_h = (float*)malloc(size);
            hist_h = (unsigned int*)malloc(bsize);
            // Check memory allocations
            if(data_h == NULL || hist_h == NULL){
                fprintf(stderr, "Failed to allocate host vectors!\n");
                exit(EXIT_FAILURE);
            }

            for(int i = 0; i < bins; i = i+1){
                hist_h[i]=0;
            }

            printf("Starting to generate data by RNG\n");
            fflush(stdout);

            // Generate an exponential decay data vector
            RandomExpDecay(data_h, N);
            printf("Finish the generaton of data\n");
            fflush(stdout);

            // start the timer
            hipEventRecord(start,0);
        
            // Allocate vectors in device memory
            hipMalloc((void**)&hist_d, bsize);
            hipMalloc((void**)&data_d, size);
        
            // Copy vectors from host memory to device memory
            hipMemcpy(data_d, data_h, size, hipMemcpyHostToDevice);
            hipMemcpy(hist_d, hist_h, bsize, hipMemcpyHostToDevice);
        
            // stop the timer
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);
        
            hipEventElapsedTime( &Intime, start, stop);
            printf("Input time for GPU: %f (ms) \n",Intime);

            // start the timer
            hipEventRecord(start,0);

            sm = threadsPerBlock * sizeof(int);
            hist_shmem <<< blocksPerGrid, threadsPerBlock, sm >>> (data_d, N, hist_d, bins, Rmin, binsize);
        
            // stop the timer
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);
        
            hipEventElapsedTime( &gputime, start, stop);
            printf("Processing time for GPU: %f (ms) \n",gputime);
            printf("GPU Gflops: %f\n",2*N/(1000000.0*gputime));

            // Copy result from device memory to host memory
            // hist_h contains the result in host memory
        
            // start the timer
            hipEventRecord(start,0);
        
            hipMemcpy(hist_h, hist_d, bsize, hipMemcpyDeviceToHost);
        
            hipFree(data_d);
            hipFree(hist_d);
        
            // stop the timer
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);
        
            hipEventElapsedTime( &Outime, start, stop);
            printf("Output time for GPU: %f (ms) \n",Outime);

            gputime_tot = Intime + gputime + Outime;
            printf("Total time for GPU: %f (ms) \n",gputime_tot);

            // save histogram in file
            FILE *out;            
            out = fopen("hist_shmem.dat","w");
        
            fprintf(out, "Histogram (GPU):\n");
            for(int i = 0; i < bins; i = i+1) {
                float x = Rmin + (i + 0.5) * binsize;         // the center of each bin
                fprintf(out,"%f %d\n", x, hist_h[i]);
            }
            fclose(out);

            /*
            Compute the CPU reference solution
             */
            
            // Allocate memory 
            hist_c = (unsigned int*)malloc(bsize);
            for(int i=0; i<bins; i++){
                hist_c[i]=0;
            }

            // start the timer
            hipEventRecord(start,0);
        
            for(int i = 0; i < N; i = i+1) {
                index = (long) ((data_h[i] - Rmin) / binsize);
                if( (index > bins-1) || (index < 0)) {
                    printf("data[%d]=%f, index=%d\n", i, data_h[i], index);
                    exit(0);
                } 
                hist_c[index]++;
            }

            // stop the timer
            hipEventRecord(stop,0);
            hipEventSynchronize(stop);

            hipEventElapsedTime( &cputime, start, stop);
            printf("Processing time for CPU: %f (ms) \n",cputime);
            printf("CPU Gflops: %f\n",2*N/(1000000.0*cputime));
            printf("Speed up of GPU = %f\n", cputime/(gputime_tot));
    
            // check histogram sum equal to the total number of data 
            int sum = 0;
            for(int i = 0; i < bins; i = i+1) {
                sum = sum + hist_c[i];
            }
            if(sum != N) {
                printf("Error, sum = %d\n",sum);
                exit(0);
            }

            // compare histograms from CPU and GPU
            for(int i = 0; i < bins; i = i+1) {
                if(hist_h[i] != hist_c[i]){
                    printf("Block Size = %d, Grid Size = %d\n", threadsPerBlock, blocksPerGrid);
                    printf("i=%d, hist_h=%d, hist_c=%d \n", i, hist_h[i], hist_c[i]);
                }
            }

            // save histogram in file
            FILE *out1;           
            out1 = fopen("hist_cpu.dat","w");
        
            fprintf(out1, "Histogram (CPU):\n");
            for(int i = 0; i < bins; i = i+1) {
                float x = Rmin + (i + 0.5) * binsize;         // the center of each bin
                fprintf(out1,"%f %d\n", x, hist_c[i]);
            }
            fclose(out1);

            // Output time result to file
            output = fopen("hist_shmem_result.txt", "a");
            fprintf(output, "%d %d %d %f %f %f %.3f\n", bins, threadsPerBlock, blocksPerGrid, gputime, gputime_tot, cputime, cputime/gputime_tot);
            fclose(output);

            free(data_h);
            free(hist_h);
            free(hist_c);
        }
    }


    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Reset the device
    hipDeviceReset();

    return 0;
}


void RandomUniform(float* data, long n)   // RNG with uniform distribution in (0,1)
{
    for(long i = 0; i < n; i++){
        data[i] = rand()/(float)RAND_MAX;
    }
}

void RandomNormal(float* data, long n)   // RNG with normal distribution, mu=0, sigma=1
{
    const float Pi = acos(-1.0);

    for(long i = 0; i < n; i++) {
        double y = (double) rand() / (float)RAND_MAX;
        double x = -log(1.0-y);
        double z = (double) rand() / (float)RAND_MAX;
        double theta = 2*Pi*z;
        data[i] = (float) (sqrt(2.0*x)*cos(theta));   
    }
}


void RandomExpDecay(float* data, long n)   // RNG with Exponential Decay
{
    for(long i = 0; i < n; i = i+1){
        double y = (double) rand() / (float) RAND_MAX;
        data[i] = (float) -log(1.0 - y);
    }
}
