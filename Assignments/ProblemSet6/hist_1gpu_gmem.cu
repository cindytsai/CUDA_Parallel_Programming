// To compute histogram with atomic operations */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


// Variables
float* data_h;          // host vectors
unsigned int* hist_h;   
float* data_d;          // device vectors
unsigned int* hist_d;
unsigned int* hist_c;   // CPU solution


// Functions
void RandomUniform(float*, long);
void RandomNormal(float*, long);
void RandomExpDecay(float*, long);


__global__ void hist_gmem(float *data, const long N, unsigned int *hist, 
                          const int bins, const float Rmin, const float binsize) 
{

    // use global memory and atomic addition
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    long stride = blockDim.x * gridDim.x;

//    if( (index > bins-1) || (index < 0)) {
//      printf("data[%d]=%f, index=%d\n",i,data[i],index);
//    }

    while (i < N) {
        int index = (int)((data[i]-Rmin) / binsize);
        atomicAdd(&hist[index], 1);
        i += stride;       // goto the next grid
    }

    __syncthreads();

}


int main(void)
{

    int gid;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    scanf("%d",&gid);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Set GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("To find the histogram of a data set (with real numbers): \n");
    long N;                     // Size of the data vector
    int bins;                   // Number of bins of the histogram
    int index;                  // index for the histogram
    float Rmin, Rmax;           // Range of the distribution of the random number
    float binsize;              // Range of each bins

    printf("Enter the size of the data vector: ");
    scanf("%ld",&N);
    printf("%ld\n",N);
    long size = N * sizeof(float);  // size of the data vector

    printf("Enter the data range [Rmin, Rmax] for the histogram: ");
    scanf("%f %f",&Rmin, &Rmax);
    printf("%f %f\n",Rmin, Rmax);
    fflush(stdout);

    printf("Enter the number of bins of the histogram: ");
    scanf("%d",&bins);
    printf("%d\n",bins);
    fflush(stdout);
    int bsize = bins * sizeof(int);   // size of the histogram
    binsize = (Rmax - Rmin) / (float)bins;
     
    data_h = (float*)malloc(size);
    hist_h = (unsigned int*)malloc(bsize);

    // Check memory allocations
    if(data_h == NULL || hist_h == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for(int i=0; i<bins; i++){
        hist_h[i]=0;
    }

    // initialize the data_h vector
    // srand(time(NULL));  // initialize the seed with the current time
    
    //Set Seed for rand() Function.
    srand(12345);

    printf("Starting to generate data by RNG\n");
    fflush(stdout);

    // Generate an exponential decay data vector
    RandomExpDecay(data_h, N);
    printf("Finish the generaton of data\n");
    fflush(stdout);

    int threadsPerBlock;
    printf("Enter the number of threads per block: ");
    scanf("%d",&threadsPerBlock);
    printf("%d\n",threadsPerBlock);
    fflush(stdout);
    if( threadsPerBlock > 1024 ) {
        printf("The number of threads per block must be less than 1024 ! \n");
        exit(0);
    }

    int blocksPerGrid;
    printf("Enter the number of blocks per grid: ");
    scanf("%d",&blocksPerGrid);
    printf("%d\n",blocksPerGrid);
    if( blocksPerGrid > 2147483647 ) {
        printf("The number of blocks must be less than 2147483647 ! \n");
        exit(0);
    }
    printf("The number of blocks is %d\n", blocksPerGrid);
    fflush(stdout);

    int CPU;
    printf("To compute the histogram with CPU (1/0) ? ");
    scanf("%d",&CPU);
    printf("%d\n",CPU);
    fflush(stdout);


    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start,0);

    // Allocate vectors in device memory

    hipMalloc((void**)&hist_d, bsize);
    hipMalloc((void**)&data_d, size);

    // Copy vectors from host memory to device memory

    hipMemcpy(data_d, data_h, size, hipMemcpyHostToDevice);
    hipMemcpy(hist_d, hist_h, bsize, hipMemcpyHostToDevice);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Intime;
    hipEventElapsedTime( &Intime, start, stop);
    printf("Input time for GPU: %f (ms) \n",Intime);

    // start the timer
    hipEventRecord(start,0);

    hist_gmem <<< blocksPerGrid, threadsPerBlock >>> (data_d, N, hist_d, bins, Rmin, binsize);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float gputime;
    hipEventElapsedTime( &gputime, start, stop);
    printf("Processing time for GPU: %f (ms) \n",gputime);
    printf("GPU Gflops: %f\n",2*N/(1000000.0*gputime));

    // Copy result from device memory to host memory
    // hist_h contains the result in host memory

    // start the timer
    hipEventRecord(start,0);

    hipMemcpy(hist_h, hist_d, bsize, hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(hist_d);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Outime;
    hipEventElapsedTime( &Outime, start, stop);
    printf("Output time for GPU: %f (ms) \n",Outime);

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n",gputime_tot);

    // save histogram in file
    FILE *out;            
    out = fopen("hist_gmem.dat","w");

    fprintf(out, "Histogram (GPU):\n");
    for(int i = 0; i < bins; i = i+1) {
        float x = Rmin + (i + 0.5) * binsize;         // the center of each bin
        fprintf(out,"%f %d\n", x, hist_h[i]);
    }
    fclose(out);

    // print histogram on screen
    printf("Histogram (GPU):\n");
    for(int i = 0; i < bins; i = i+1) {
        float x = Rmin + (i + 0.5) * binsize;         // the center of each bin
        printf("%f %d\n", x, hist_h[i]);
    }

    if(CPU==0) {
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipDeviceReset();
        free(data_h);
        free(hist_h);
        return 0;
    }

    // To compute the CPU reference solution 

    hist_c = (unsigned int*)malloc(bsize);
    for(int i=0; i<bins; i++){
        hist_c[i]=0;
    }

    // start the timer
    hipEventRecord(start,0);

    for(int i = 0; i < N; i = i+1) {
        index = (long) ((data_h[i] - Rmin) / binsize);
        if( (index > bins-1) || (index < 0)) {
            printf("data[%d]=%f, index=%d\n", i, data_h[i], index);
            exit(0);
        } 
        hist_c[index]++;
    }

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",2*N/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/(gputime_tot));

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check histogram sum equal to the total number of data 

    int sum = 0;
    for(int i = 0; i < bins; i = i+1) {
        sum = sum + hist_c[i];
    }
    if(sum != N) {
        printf("Error, sum = %d\n",sum);
        exit(0);
    }

    // compare histograms from CPU and GPU

    for(int i = 0; i < bins; i = i+1) {
        if(hist_h[i] != hist_c[i]){
            printf("i=%d, hist_h=%d, hist_c=%d \n", i, hist_h[i], hist_c[i]);
        } 
    }

    // save histogram in file
    FILE *out1;           
    out1 = fopen("hist_cpu.dat","w");

    fprintf(out1, "Histogram (CPU):\n");
    for(int i = 0; i < bins; i = i+1) {
        float x = Rmin + (i + 0.5) * binsize;         // the center of each bin
        fprintf(out1,"%f %d\n", x, hist_c[i]);
    }
    fclose(out1);

    // print the histogram on screen
    printf("Histogram (CPU):\n");
    for(int i = 0; i < bins; i =i+1) {
        float x = Rmin + (i + 0.5) * binsize;         // the center of each bin
        printf("%f %d\n", x, hist_c[i]);
    }

    hipDeviceReset();

    free(data_h);
    free(hist_h);
    free(hist_c);

    return 0;
}

void RandomUniform(float* data, long n)   // RNG with uniform distribution in (0,1)
{
    for(long i = 0; i < n; i++){
        data[i] = rand()/(float)RAND_MAX;
    }
}

void RandomNormal(float* data, long n)   // RNG with normal distribution, mu=0, sigma=1
{
    const float Pi = acos(-1.0);

    for(long i = 0; i < n; i++) {
        double y = (double) rand() / (float)RAND_MAX;
        double x = -log(1.0-y);
        double z = (double) rand() / (float)RAND_MAX;
        double theta = 2*Pi*z;
        data[i] = (float) (sqrt(2.0*x)*cos(theta));   
    }
}


void RandomExpDecay(float* data, long n)   // RNG with Exponential Decay
{
    for(long i = 0; i < n; i = i+1){
        double y = (double) rand() / (float) RAND_MAX;
        data[i] = (float) -log(1.0 - y);
    }
}
