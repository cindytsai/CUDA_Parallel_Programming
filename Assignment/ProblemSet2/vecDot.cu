// Vector Dot Product A.B
// compile with the following command:
//
// (for GTX970)
// nvcc -arch=compute_52 -code=sm_52,sm_52 -O2 -m64 -o vecAdd vecAdd.cu
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O2 -m64 -o vecAdd vecAdd.cu


// Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Variables
float* h_A;   // host vectors
float* h_B;
float* h_C;
float* d_A;   // device vectors
float* d_B;
float* d_C;

// Functions
void RandomInit(float*, int);

// Device code
__global__ void VecDot(const float* A, const float* B, float* C, int N)
{
    extern __shared__ float cache[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0;  // register for each thread
    while (i < N) {
        temp += A[i]*B[i];
        i += blockDim.x*gridDim.x;  
    }
   
    cache[cacheIndex] = temp;   // set the cache value 

    __syncthreads();

    // perform parallel reduction, threadsPerBlock must be 2^m

    int ib = blockDim.x/2;
    while (ib != 0) {
      if(cacheIndex < ib)
        cache[cacheIndex] += cache[cacheIndex + ib]; 

      __syncthreads();

      ib /=2;
    }
    
    if(cacheIndex == 0)
      C[blockIdx.x] = cache[0];

}

// Host code

int main(void)
{

    int gid;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    printf("Enter the GPU ID: ");
    scanf("%d",&gid);
    printf("%d\n", gid);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Set GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("Vector Dot Product: A.B\n");
    int N;

    printf("Enter the size of the vectors: ");
    scanf("%d",&N);        
    printf("%d\n",N);        

    // Set the sizes of threads and blocks

    int threadsPerBlock;
    printf("Enter the number (2^m) of threads per block: ");
    scanf("%d",&threadsPerBlock);
    printf("%d\n",threadsPerBlock);
    if( threadsPerBlock > 1024 ) {
      printf("The number of threads per block must be less than 1024 ! \n");
      exit(0);
    }

//    int blocksPerGrid = (N + threadsPerBlock - 1)/threadsPerBlock;
//    printf("The number of blocks per grid:%d\n",blocksPerGrid);
 
    int blocksPerGrid;
    printf("Enter the number of blocks per grid: ");
    scanf("%d",&blocksPerGrid);
    printf("%d\n",blocksPerGrid);

    if( blocksPerGrid > 2147483647 ) {
      printf("The number of blocks must be less than 2147483647 ! \n");
      exit(0);
    }

    // Allocate input vectors h_A and h_B in host memory

    int size = N * sizeof(float);
    int sb = blocksPerGrid * sizeof(float);

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(sb);     // contains the result of dot-product from each block
    
    // Initialize input vectors

    RandomInit(h_A, N);
    RandomInit(h_B, N);


    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start,0);

    // Allocate vectors in device memory

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, sb);

    // Copy vectors from host memory to device memory

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Intime;
    hipEventElapsedTime( &Intime, start, stop);
    printf("Input time for GPU: %f (ms) \n",Intime);

    // start the timer
    hipEventRecord(start,0);

    int sm = threadsPerBlock*sizeof(float);
    VecDot <<< blocksPerGrid, threadsPerBlock, sm >>>(d_A, d_B, d_C, N);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float gputime;
    hipEventElapsedTime( &gputime, start, stop);
    printf("Processing time for GPU: %f (ms) \n",gputime);
    printf("GPU Gflops: %f\n",(2*N-1)/(1000000.0*gputime));
    
    // Copy result from device memory to host memory
    // h_C contains the result of each block in host memory

    // start the timer
    hipEventRecord(start,0);

    hipMemcpy(h_C, d_C, sb, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    float h_G=0.0;
    for(int i = 0; i < blocksPerGrid; i++) 
      h_G += h_C[i];
    

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Outime;
    hipEventElapsedTime( &Outime, start, stop);
    printf("Output time for GPU: %f (ms) \n",Outime);

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n",gputime_tot);

    // start the timer
    hipEventRecord(start,0);

    // to compute the reference solution

    double h_D=0.0;       
    for(int i = 0; i < N; i++) 
      h_D += (double) h_A[i]*h_B[i];
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",(2*N-1)/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/(gputime_tot));

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result

    printf("Check result:\n");
    float diff = abs( (h_D - h_G)/h_D );
    printf("|(h_G - h_D)/h_D|=%20.15e\n",diff);
    printf("h_G =%20.15e\n",h_G);
    printf("h_D =%20.15e\n",h_D);

    free(h_A);
    free(h_B);
    free(h_C);

    hipDeviceReset();
}


// Allocates an array with random float entries in (-1,1)
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = 2.0*rand()/(float)RAND_MAX - 1.0;
}



