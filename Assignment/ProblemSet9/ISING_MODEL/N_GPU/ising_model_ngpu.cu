#include "hip/hip_runtime.h"
/*This version synchronize the N_GPU calculation. */
# include <stdio.h>
# include <stdlib.h>
# include <string.h>
# include <math.h>
# include <omp.h>
# include <hip/hip_runtime.h>
# include <hiprand/hiprand_kernel.h>
# include <gsl/gsl_sf_ellint.h>
# include <gsl/gsl_rng.h>
# define _USE_MATH_DEFINES

__constant__ double exp_update0_dev[5];
__constant__ double exp_update1_dev[2];
__device__ int UPDATE_SPIN_GPU (int, int, int, double);

void EXACT_EM (double*, double*);
void UPDATE_MATRIX(double*, double*);
void UPDATE_SPIN(int, int, int, int, int, int*, double*, gsl_rng*);
void ISING_CPU (void);

__global__ void SET_UP_KERNEL (long seed, hiprandState *state)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	hiprand_init (seed, index, 0, &state[index]);
}

__global__ void INITIALIZE_COLD (int *d_spin)
{
	int index_x = threadIdx.x + blockIdx.x*blockDim.x;
	int index_y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = index_x + index_y*blockDim.x*gridDim.x;

	d_spin[index] = 1;
}

__global__ void INITIALIZE_HOT (int *d_spin, hiprandState *state)
{
	int index_x = threadIdx.x + blockIdx.x*blockDim.x;
	int index_y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = index_x + index_y*blockDim.x*gridDim.x;
	hiprandState local = state[index];
	double x = hiprand_uniform(&local);

	if (x<0.5)
		d_spin[index] = 1;
	else
		d_spin[index] = -1;
//	printf("%d\t%.4f\n", index, x);
	state[index] = local;
}

__global__ void DOUBLE_CHECKERBOARD_SETUP(int *oTiB, int *eTiB)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Tx = blockDim.x;
	int thread_index = tx + ty*Tx;
	int cache_index = (tx+1) + (ty+1)*(Tx+2);
	int cache_index_r = (tx+2) + (ty+1)*(Tx+2);
	int cache_index_l = tx + (ty+1)*(Tx+2);
	int cache_index_u = (tx+1) + (ty+2)*(Tx+2);
	int cache_index_d = (tx+1) + ty*(Tx+2);
	// odd threads
	if (threadIdx.x%2!=threadIdx.y%2)
	{
		oTiB[5*(thread_index/2)] = cache_index;
		oTiB[5*(thread_index/2)+1] = cache_index_r;
		oTiB[5*(thread_index/2)+2] = cache_index_l;
		oTiB[5*(thread_index/2)+3] = cache_index_u;
		oTiB[5*(thread_index/2)+4] = cache_index_d;
	}
	//
	// even threads
	else if (threadIdx.x%2==threadIdx.y%2)
	{
		eTiB[5*(thread_index/2)] = cache_index;
		eTiB[5*(thread_index/2)+1] = cache_index_r;
		eTiB[5*(thread_index/2)+2] = cache_index_l;
		eTiB[5*(thread_index/2)+3] = cache_index_u;
		eTiB[5*(thread_index/2)+4] = cache_index_d;
//			printf("%d\t%d\t%d\t\n", block_index, cache_index_d, eTioB[block_index/2][5*(thread_index/2)+4]);
	}
	//
}

__global__ void ISING_ODD_BLOCK(int interval_measure, double B, int *d_spin, int *d_spin_l, int *d_spin_r, int *d_spin_d, int *d_spin_u, hiprandState *state, int *oTiB, int *eTiB)
{
	extern __shared__ int cache[];
	int lattice_center_odd, lattice_center_even;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Tx = blockDim.x;
	int Ty = blockDim.y;
	int thread_index = tx + ty*Tx;
	int block_index = blockIdx.x + gridDim.x*blockIdx.y;
	int Bx = 2*gridDim.x;
	int By = gridDim.y;
	int Nx = 2*Tx*Bx;
	int Ny = Ty*By;
	int bx = (2*block_index)%Bx;
	int by = (2*block_index)/Bx;
	//for odd block
	int parity = (bx+by+1)%2;
	bx += parity;
	// mount odd site in each odd block to shared
	int x = (2*thread_index)%(2*Tx);
	int y = (2*thread_index)/(2*Tx);
	parity = (x+y+1)%2;
	x += parity;
	int cache_index = (x+1) + (y+1)*(2*Tx+2);
	x += 2*Tx*bx;
	y += Ty*by;
	int lattice_index = x + y*Nx;
	lattice_center_odd = lattice_index;  // to copy the updated spin from shared to d_spin
	cache[cache_index] = d_spin[lattice_index];
//	if (block_index==0)
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
	//
	// mount even site in each block to shared
	x = (2*thread_index)%(2*Tx);
	y = (2*thread_index)/(2*Tx);
	parity = (x+y)%2;
	x += parity;
	cache_index = (x+1) + (y+1)*(2*Tx+2);
	x += 2*Tx*bx;
	y += Ty*by;
	lattice_index = x + y*Nx;
	lattice_center_even = lattice_index;  // to copy the updated spin from shared to d_spin
	cache[cache_index] = d_spin[lattice_index];
//	if (block_index==2047)
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
	//
	// mount the boundary to shared
	if (Tx==1)
	{
		// left
		x = 0;
		y = ty; 
		cache_index = (y+1)*(2*Tx+2);
		x += 2*Tx*bx;
		y += Ty*by;
		lattice_index = (x-1+Nx)%Nx + y*Nx;
		if (x==0)
			cache[cache_index] = d_spin_l[lattice_index];
		else
			cache[cache_index] = d_spin[lattice_index];
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		// right
		x = 1;
		y = ty;
		cache_index = (x+2) + (y+1)*(2*Tx+2);
		x += 2*Tx*bx;
		y += Ty*by;
		lattice_index = (x+1)%Nx + y*Nx;
		if (x==Nx-1)
			cache[cache_index] = d_spin_r[lattice_index];
		else
			cache[cache_index] = d_spin[lattice_index];
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		// down
		if (ty==0)
		{
			for (int i=0; i<2; i++)
			{
				x = i;
				y = 0;
				cache_index = x+1;
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y-1+Ny)%Ny*Nx;
				if (y==0)
					cache[cache_index] = d_spin_d[lattice_index];
				else
					cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
		// up
		else if (ty==Ty-1)
		{
			for (int i=0; i<2; i++)
			{
				x = i;
				y = Ty-1;
				cache_index = (x+1) + (y+2)*(2*Tx+2);
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y+1)%Ny*Nx;
				if (y==Ny-1)
					cache[cache_index] = d_spin_u[lattice_index];
				else
					cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
	}
	else
	{
		// left
		if (tx==0)
		{
			x = 0;
			y = ty;
			cache_index = (y+1)*(2*Tx+2);
			x += 2*Tx*bx;
			y += Ty*by;
			lattice_index = (x-1+Nx)%Nx + y*Nx;
			if (x==0)
				cache[cache_index] = d_spin_l[lattice_index];
			else
				cache[cache_index] = d_spin[lattice_index];
//			printf("%d\t%d\n", bx*Bx, by*By);
//			printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		}
		// right
		else if (tx==Tx-1)
		{
			x = 2*Tx-1;
			y = ty;
			cache_index = (x+2) + (y+1)*(2*Tx+2);
			x += 2*Tx*bx;
			y += Ty*by;
			lattice_index = (x+1)%Nx + y*Nx;
			if (x==Nx-1)
				cache[cache_index] = d_spin_r[lattice_index];
			else
				cache[cache_index] = d_spin[lattice_index];
//			printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		}
		// down
		if (ty==0)
		{
			for (int i=0; i<2; i++)
			{
				x = 2*tx+i;
				y = 0;
				cache_index = x+1;
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y-1+Ny)%Ny*Nx;
				if (y==0)
					cache[cache_index] = d_spin_d[lattice_index];
				else
					cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
		// up
		else if (ty==Ty-1)
		{
			for (int i=0; i<2; i++)
			{
				x = 2*tx+i;
				y = Ty-1;
				cache_index = (x+1) + (y+2)*(2*Tx+2);
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y+1)%Ny*Nx;
				if (y==Ny-1)
					cache[cache_index] = d_spin_u[lattice_index];
				else
					cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
	}
	__syncthreads();
	//
	hiprandState local = state[threadIdx.x+blockDim.x*blockIdx.x + blockDim.x*gridDim.x*(threadIdx.y+blockDim.y*blockIdx.y)];
	int center, right, left, up, down;
	int spin_r, spin_l, spin_u, spin_d;
	int old_spin, new_spin, spin_around;
	int delta_E;
	// spin update
		// odd threads
	center = oTiB[5*thread_index];
	right = oTiB[5*thread_index+1];
	left = oTiB[5*thread_index+2];
	up = oTiB[5*thread_index+3];
	down = oTiB[5*thread_index+4];

	for (int iter=0; iter<interval_measure; iter++)
	{
		spin_r = cache[right];
		spin_l = cache[left];
		spin_u = cache[up];
		spin_d = cache[down];
		old_spin = cache[center];
		new_spin = -old_spin;
		spin_around = spin_r + spin_l + spin_u  + spin_d;
		if (spin_around%2!=0)
			printf("Odd thread Odd block: %d\t%d\t%d\t%d\t%d\n", center, spin_r, spin_l, spin_u, spin_d);
		delta_E = (old_spin-new_spin)*(spin_around+B);
		if (delta_E<=0)
			cache[center] = new_spin;
		else
		{
			double x = hiprand_uniform(&local);
			cache[center] = UPDATE_SPIN_GPU(old_spin, new_spin, spin_around, x);
//			if (thread_index==0)
//				printf("%.4f\n", x);
		}
	}
	d_spin[lattice_center_odd] = cache[center];
	__syncthreads();
		//
		// even threads;
	center = eTiB[5*thread_index];
	right = eTiB[5*thread_index+1];
	left = eTiB[5*thread_index+2];
	up = eTiB[5*thread_index+3];
	down = eTiB[5*thread_index+4];

	for (int iter=0; iter<interval_measure; iter++)
	{
		spin_r = cache[right];
		spin_l = cache[left];
		spin_u = cache[up];
		spin_d = cache[down];
		old_spin = cache[center];
		new_spin = -old_spin;
		spin_around = spin_r + spin_l + spin_u  + spin_d;
		if (spin_around%2!=0)
			printf("Even thread Odd block: %d\t%d\t%d\t%d\t%d\n", center, spin_r, spin_l, spin_u, spin_d);
		delta_E = (old_spin-new_spin)*(spin_around+B);
		if (delta_E<=0)
			cache[center] = new_spin;
		else
		{
			double x = hiprand_uniform(&local);
			cache[center] = UPDATE_SPIN_GPU(old_spin, new_spin, spin_around, x);
		}
	}
	d_spin[lattice_center_even] = cache[center];
	state[threadIdx.x+blockDim.x*blockIdx.x + blockDim.x*gridDim.x*(threadIdx.y+blockDim.y*blockIdx.y)]=local;
	__syncthreads();
		//
}

__global__ void ISING_EVEN_BLOCK(int interval_measure, double B, int *d_spin, int *d_spin_l, int *d_spin_r, int *d_spin_d, int *d_spin_u, hiprandState *state, int *oTiB, int *eTiB)
{
	extern __shared__ int cache[];
	int lattice_center_odd, lattice_center_even;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Tx = blockDim.x;
	int Ty = blockDim.y;
	int thread_index = tx + ty*Tx;
	int block_index = blockIdx.x + gridDim.x*blockIdx.y;
	int Bx = 2*gridDim.x;
	int By = gridDim.y;
	int Nx = 2*Tx*Bx;
	int Ny = Ty*By;
	int bx = (2*block_index)%Bx;
	int by = (2*block_index)/Bx;
	// for even block
	int parity = (bx+by)%2;
	bx += parity;
	// mount odd site in each odd block to shared
	int x = (2*thread_index)%(2*Tx);
	int y = (2*thread_index)/(2*Tx);
	parity = (x+y+1)%2;
	x += parity;
	int cache_index = (x+1) + (y+1)*(2*Tx+2);
	x += 2*Tx*bx;
	y += Ty*by;
	int lattice_index = x + y*Nx;
	lattice_center_odd = lattice_index; // to copy the updated spin from shared to d_spin
	cache[cache_index] = d_spin[lattice_index];
//	if (block_index==3)
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
	//
	// mount even site in each block to shared
	x = (2*thread_index)%(2*Tx);
	y = (2*thread_index)/(2*Tx);
	parity = (x+y)%2;
	x += parity;
	cache_index = (x+1) + (y+1)*(2*Tx+2);
	x += 2*Tx*bx;
	y += Ty*by;
	lattice_index = x + y*Nx;
	lattice_center_even = lattice_index; // to copy the updated spin from shared to d_spin
	cache[cache_index] = d_spin[lattice_index];
//	if (block_index==2)
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
	//
	// mount the boundary to shared
	if (Tx==1)
	{
		// left
		x = 0;
		y = ty; 
		cache_index = (y+1)*(2*Tx+2);
		x += 2*Tx*bx;
		y += Ty*by;
		lattice_index = (x-1+Nx)%Nx + y*Nx;
		if (x==0)
			cache[cache_index] = d_spin_l[lattice_index];
		else
			cache[cache_index] = d_spin[lattice_index];
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		// right
		x = 1;
		y = ty;
		cache_index = (x+2) + (y+1)*(2*Tx+2);
		x += 2*Tx*bx;
		y += Ty*by;
		lattice_index = (x+1)%Nx + y*Nx;
		if (x==Nx-1)
			cache[cache_index] = d_spin_r[lattice_index];
		else
			cache[cache_index] = d_spin[lattice_index];
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		// down
		if (ty==0)
		{
			for (int i=0; i<2; i++)
			{
				x = i;
				y = 0;
				cache_index = x+1;
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y-1+Ny)%Ny*Nx;
				if (y==0)
					cache[cache_index] = d_spin_d[lattice_index];
				else
					cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
		// up
		else if (ty==Ty-1)
		{
			for (int i=0; i<2; i++)
			{
				x = i;
				y = Ty-1;
				cache_index = (x+1) + (y+2)*(2*Tx+2);
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y+1)%Ny*Nx;
				if (y==Ny-1)
					cache[cache_index] = d_spin_u[lattice_index];
				else
					cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
	}
	else
	{
		// left
		if (tx==0)
		{
			x = 0;
			y = ty;
			cache_index = (y+1)*(2*Tx+2);
			x += 2*Tx*bx;
			y += Ty*by;
			lattice_index = (x-1+Nx)%Nx + y*Nx;
			if (x==0)
				cache[cache_index] = d_spin_l[lattice_index];
			else
				cache[cache_index] = d_spin[lattice_index];
//			printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		}
		// right
		else if (tx==Tx-1)
		{
			x = 2*Tx-1;
			y = ty;
			cache_index = (x+2) + (y+1)*(2*Tx+2);
			x += 2*Tx*bx;
			y += Ty*by;
			lattice_index = (x+1)%Nx + y*Nx;
			if (x==Nx-1)
				cache[cache_index] = d_spin_r[lattice_index];
			else
				cache[cache_index] = d_spin[lattice_index];
//			printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		}
		// down
		if (ty==0)
		{
			for (int i=0; i<2; i++)
			{
				x = 2*tx+i;
				y = 0;
				cache_index = x+1;
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y-1+Ny)%Ny*Nx;
				if (y==0)
					cache[cache_index] = d_spin_d[lattice_index];
				else
					cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
		// up
		else if (ty==Ty-1)
		{
			for (int i=0; i<2; i++)
			{
				x = 2*tx+i;
				y = Ty-1;
				cache_index = (x+1) + (y+2)*(2*Tx+2);
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y+1)%Ny*Nx;
				if (y==Ny-1)
					cache[cache_index] = d_spin_u[lattice_index];
				else
					cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
	}
	__syncthreads();
	//
	hiprandState local = state[threadIdx.x+blockDim.x*blockIdx.x + blockDim.x*gridDim.x*(threadIdx.y+blockDim.y*blockIdx.y)];
	int center, right, left, up, down;
	int spin_r, spin_l, spin_u, spin_d;
	int old_spin, new_spin, spin_around;
	int delta_E;
	// spin update
		// odd threads
	center = oTiB[5*thread_index];
	right = oTiB[5*thread_index+1];
	left = oTiB[5*thread_index+2];
	up = oTiB[5*thread_index+3];
	down = oTiB[5*thread_index+4];

	for (int iter=0; iter<interval_measure; iter++)
	{
		spin_r = cache[right];
		spin_l = cache[left];
		spin_u = cache[up];
		spin_d = cache[down];
		old_spin = cache[center];
		new_spin = -old_spin;
		spin_around = spin_r + spin_l + spin_u  + spin_d;
		if (spin_around%2!=0)
			printf("Odd thread Even block: %d\t%d\t%d\t%d\t%d\n", center, spin_r, spin_l, spin_u, spin_d);
		delta_E = (old_spin-new_spin)*(spin_around+B);
		if (delta_E<=0)
			cache[center] = new_spin;
		else
		{
			double x = hiprand_uniform(&local);
			cache[center] = UPDATE_SPIN_GPU(old_spin, new_spin, spin_around, x);
		}
	}
	d_spin[lattice_center_odd] = cache[center];
	__syncthreads();
		//
		// even threads;
	center = eTiB[5*thread_index];
	right = eTiB[5*thread_index+1];
	left = eTiB[5*thread_index+2];
	up = eTiB[5*thread_index+3];
	down = eTiB[5*thread_index+4];

	for (int iter=0; iter<interval_measure; iter++)
	{
		spin_r = cache[right];
		spin_l = cache[left];
		spin_u = cache[up];
		spin_d = cache[down];
		old_spin = cache[center];
		new_spin = -old_spin;
		spin_around = spin_r + spin_l + spin_u  + spin_d;
		if (spin_around%2!=0)
		{
			printf("Even threads Even block: %d\t%d\t%d\t%d\t%d\t%d\n", block_index, center, spin_r, spin_l, spin_u, spin_d);
//			printf("%d\n", eTieB[block_index][5*thread_index+4]);
		}
		delta_E = (old_spin-new_spin)*(spin_around+B);
		if (delta_E<=0)
			cache[center] = new_spin;
		else
		{
			double x = hiprand_uniform(&local);
			cache[center] = UPDATE_SPIN_GPU(old_spin, new_spin, spin_around, x);
		}
	}
	d_spin[lattice_center_even] = cache[center];
	state[threadIdx.x+blockDim.x*blockIdx.x + blockDim.x*gridDim.x*(threadIdx.y+blockDim.y*blockIdx.y)]=local;
	__syncthreads();
		//
}

__global__ void  ISING_MEASUREMENT (double B, int *d_spin, int *d_spin_r, int *d_spin_u, int *d_M, int *d_E)
{
	extern __shared__ int cache[];
	int layer = blockDim.x*blockDim.y/2;
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	int Nx = blockDim.x*gridDim.x;
	int Ny = blockDim.y*gridDim.y;
	int index = x + Nx*y;
	int index_r = (x+1)%Nx + Nx*y;
	int index_u = x + Nx*((y+1)%Ny);
	int cache_index = threadIdx.x + blockDim.x*threadIdx.y;
	int block_index = blockIdx.x + gridDim.x*blockIdx.y;
	int shift = blockDim.x*blockDim.y;
	int spin_r, spin_u;
	if (x==Nx-1)
		spin_r = d_spin_r[index_r];
	else
		spin_r = d_spin[index_r];
	if (y==Ny-1)
		spin_u = d_spin_u[index_u];
	else
		spin_u = d_spin[index_u];
	cache[cache_index] = d_spin[index];
	cache[cache_index+shift] = -d_spin[index]*(spin_r+spin_u+B);
	__syncthreads();

	while (layer>0)
	{
		if (cache_index<layer)
		{
			cache[cache_index] += cache[cache_index+layer];
			cache[cache_index+shift] += cache[cache_index+shift+layer];
		}
		layer /= 2;
		__syncthreads();
	}	
	if (cache_index==0)
	{
		d_M[block_index] = cache[0];
		d_E[block_index] = cache[shift];
	}
}

__global__ void CONSTANT_MEMORY_TEST ()
{
//	printf("Do constant memory test.\n");
	for (int i=0; i<5; i++)
		printf("%.4f\t", exp_update0_dev[i]);
	printf("\n");
	for (int i=0; i<2; i++)
		printf("%.4f\t", exp_update1_dev[i]);
	printf("\n");
}

__global__ void RANDOM_NUMBER_TEST (int iter, hiprandState *state)
{
	int index_x = threadIdx.x + blockIdx.x*blockDim.x;
	int index_y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = index_x + index_y*blockDim.x*gridDim.x;
	hiprandState local = state[index];
	
	int N = 1;
	for (int i=0; i<N; i++)
		printf("%d\t%.4f\n", iter, hiprand_uniform(&local));
	state[threadIdx.x+blockDim.x*blockIdx.x] = local;
}

char ini;
int N_thermal, N_measure, interval_measure, interval_save, interval_display;
int Nx, Ny, N_site;
double T, B;
double Tc, E_exact, M_exact;
int *h_spin;
long seed;
gsl_rng *rng;

int main(void)
{
	char mode;
	int size_site, size_site_GPU, m, n;
	int N_GPU_x, N_GPU_y, N_GPU, tx, ty, bx, by, N_block, N_thread, cpu_thread_id;
	float gpu_time, total_time, cpu_time;
	int *gid;
	long *seed_GPU;
	hipEvent_t start, stop;
	
	puts("Simulate the 2D Ising model by N GPU.\n");
	puts("Set the number of GPU in x any direction ( N_GPU_x,N_GPU_y ).");
	scanf("%d %d", &N_GPU_x, &N_GPU_y);
	printf("The number of GPU in x and y direction is is ( %d,%d ) .\n", N_GPU_x, N_GPU_y);
	N_GPU = N_GPU_x*N_GPU_y;
	puts("Set the lattice size ( Nx,Ny ) for the 2D lattice (Nx/N_GPU_x and Ny/N_GPU_y must be even to apply checkboard scheme with periodic boundary condition).");
	scanf("%d %d", &Nx, &Ny);
	if ((Nx/N_GPU_x)%2!=0 || (Ny/N_GPU_y)%2!=0)
	{
		puts("Nx/N_GPU_x and Ny/N_GPU_y must be even! Exit!");
		exit(1);
	}
	N_site = Nx*Ny;
	printf("The lattice size is ( %d,%d ) .\n", Nx, Ny);

	size_site = N_site*sizeof(int);
	h_spin = (int*)malloc(size_site);

	puts("Set the temperature for the simulation in unit of J.");
	scanf("%lf", &T);
	if (T<0)
	{
		puts("Temperature must be positive! Exit!");
		exit(1);
	}
	printf("The temperature is %.6f J .\n", T);
	puts("Set the magnetic field for the simulation in unit of J.");
	scanf("%lf", &B);
	if (B<0)
	{
		puts("Field strength must be positive! Exit!");
		exit(1);
	}
	printf("The magnetic field is %.6f J.\n", B);
	puts("Choose cold/hot start (c/h) .");
	scanf("%c", &ini);  // absorb the \n
	scanf("%c", &ini);
	if ( ini=='c' )
		puts("Cold start is chosen.");
	else if ( ini=='h' )
		puts("Hot start is chosen");
	else
	{
		puts("Must enter c or h! Exit!");
		exit(1);
	}
	printf("\n");
	puts("Enter the number of measurement(s)");
	scanf("%d", &N_measure);
	printf("The number of measurement(s) is %d .\n", N_measure);
	puts("Enter the interval for measurement.");
	scanf("%d", &interval_measure);
	printf("The interval for measurement is %d .\n", interval_measure);
	puts("Enter the number of iterations for thermalization (must be divisibe by interval_measure).");
	scanf("%d", &N_thermal);
	if ( N_thermal%interval_measure!=0 )
	{
		puts("N_thermal is indivisible by interval_measure. Exit!");
		exit(1);
	}
	printf("The number of iterations for thermalization is %d .\n", N_thermal);
	puts("Enter the interval for display (must be divisible by interval_measure).");
	scanf("%d", &interval_display);
	if ( interval_display%interval_measure!=0 )
	{
		puts("interval_display is indivisible by interval_measure. Exit!");
		exit(1);
	}
	printf("The interval for display is %d .\n", interval_display);
	puts("Enter the interval for saving (must be divisible by interval_measure).");
	scanf("%d", &interval_save);
	if ( interval_save%interval_measure!=0 )
	{
		puts("N_thermal is indivisible by interval_measure. Exit!");
		exit(1);
	}
	printf("The interval for saving is %d .\n", interval_save);

	puts("Select the mode for simulation GPU/CPU/Both (g/c/b) .");
	scanf("%c", &mode);  // absorb the \n
	scanf("%c", &mode);
	if ( mode=='g' )
		puts("Do GPU simulation.");
	else if ( mode=='c' )
		puts("Do CPU simulation.");
	else if ( mode=='b' )
		puts("Do both GPU and CPU simulation.");
	else
	{
		puts("Must enter g, c or b! Exit!");
		exit(1);
	}
	if (mode=='c')
	{
		puts("Set the CPU seed for random number generator.");
		scanf("%ld", &seed);
		printf("The CPU seed is set to be %ld .\n", seed);
	}
	else
	{
		printf("\n");
		seed_GPU = (long *)malloc(N_GPU*sizeof(long));
		gid = (int*)malloc(N_GPU*sizeof(int));

		for (int i=0; i<N_GPU; i++)
		{
			printf("Set the seed for #%d GPU.\n", i+1);
			scanf("%ld", &seed_GPU[i]);
			printf("The seed for #%d GPU is %d .\n", i+1, seed_GPU[i]);
		}

		for (int i=0; i<N_GPU; i++)
		{
			printf("Set the GPU ID for #%d GPU.", i+1);
			scanf("%d", &gid[i]);
			printf("The GPU ID for #%d GPU is %d .\n", i+1, gid[i]);
		}

		puts("Set the value m, n such that the theads per block is ( tx,ty ) = (2^m,2^n) (Nx/N_GPU must be divisible by tx and Ny/N_GPU must be divisible by ty).");
		scanf("%d %d", &m, &n);
		if ( m<1 )
		{
			puts("m must be positive! Exit!");
			exit(1);
		}
		else if ( n<1 )
		{
			puts("n must be positive! Exit!");
			exit(1);
		}
		tx = (int)pow(2.0, m);
		ty = (int)pow(2.0, n);
		if ( (Nx/N_GPU_x)%tx!=0 )
		{
			puts("Nx/N_GPU_x must be divisible by threads per block tx! Exit!");
			printf("%d\t%d\n", tx, (Nx/N_GPU_x)%tx);
			exit(1);
		}
		if ( (Ny/N_GPU_y)%ty!=0 )
		{
			puts("Ny/N_GPU_y must be divisible by threads per block ty! Exit!");
			exit(1);
		}
		printf("Threads per block is ( %d,%d ) .\n", tx, ty);
		puts("Use only one grids to accomdate the lattice so blocks per grid is auto-matically set.");
		bx = Nx/N_GPU_x/tx;
		if (bx%2!=0)
		{
			puts("bx must be even to apply double checkboard scheme for periodic boundary condition!");
			exit(1);
		}
		by = Ny/N_GPU_y/ty;
		if (by%2!=0)
		{
			puts("by must be even to apply double checkboard scheme for periodic boundary condition!");
			exit(1);
		}
		printf("Blocks per grid is ( %d,%d ) .\n", bx, by);
		printf("\n");

		int size_block;
		int size_shared;
		int size_shared_measure;
		int to_save = interval_save/interval_measure;
		int to_display = interval_display/interval_measure;
		double M_mean, E_mean, M_sigma, E_sigma;
		int *M_GPU, *E_GPU;
		int *h_M;
		int *h_E;
		double *M_save;
		double *E_save;
		int **gid_P2P;
		int **d_M_GPU;
		int **d_E_GPU;
		int **d_spin_GPU;
		hiprandState **devstate_GPU;
		int **odd_T_in_B_GPU, **even_T_in_B_GPU;
		FILE *output = fopen("spin_config_gpu.txt", "w");
		FILE *output2 = fopen("M_and_E_gpu.txt", "w");
		dim3 TpB (tx, ty);
		dim3 BpG (bx, by);
		N_block = bx*by;
		N_thread = tx*ty;
		size_block = N_block*sizeof(int);
		size_shared = (tx+2)*(ty+2)*sizeof(int);
		size_shared_measure = 2*N_thread*sizeof(int); // to calculate both M and E;
		size_site_GPU = size_site/N_GPU;
		M_GPU = (int *)malloc(N_GPU*sizeof(int));
		E_GPU = (int *)malloc(N_GPU*sizeof(int));
		h_M = (int *)malloc(N_GPU*size_block);
		h_E = (int *)malloc(N_GPU*size_block);
		M_save = (double *)malloc(N_measure*sizeof(double));
		E_save = (double *)malloc(N_measure*sizeof(double));
		gid_P2P = (int **)malloc(N_GPU*sizeof(int *));
		d_M_GPU = (int **)malloc(N_GPU*sizeof(int *));
		d_E_GPU = (int **)malloc(N_GPU*sizeof(int *));
		d_spin_GPU = (int **)malloc(N_GPU*sizeof(int *));
		devstate_GPU = (hiprandState **)malloc(N_GPU*sizeof(hiprandState *));
		odd_T_in_B_GPU = (int **)malloc(N_GPU*sizeof(int *));
		even_T_in_B_GPU = (int **)malloc(N_GPU*sizeof(int *));

		omp_set_num_threads(N_GPU);
		#pragma omp parallel private(cpu_thread_id)
		// OpenMP starts
		{
			int gpu_id_x, gpu_id_y;
			cpu_thread_id = omp_get_thread_num();
			hipSetDevice(gid[cpu_thread_id]);

			gid_P2P[cpu_thread_id] = (int *)malloc(5*sizeof(int));
			gpu_id_x = cpu_thread_id%N_GPU_x;
			gpu_id_y = cpu_thread_id/N_GPU_x;
			gid_P2P[cpu_thread_id][0] = (gpu_id_x-1+N_GPU_x)%N_GPU_x + gpu_id_y*N_GPU_x;  // gid at left
			gid_P2P[cpu_thread_id][1] = (gpu_id_x+1)%N_GPU_x + gpu_id_y*N_GPU_x;  // gid at right
			gid_P2P[cpu_thread_id][2] = gpu_id_x + (gpu_id_y-1+N_GPU_y)%N_GPU_y*N_GPU_x;  // gid at down
			gid_P2P[cpu_thread_id][3] = gpu_id_x + (gpu_id_y+1)%N_GPU_y*N_GPU_x;  // gid at up
			gid_P2P[cpu_thread_id][4] = gpu_id_x*(Nx/N_GPU_x) + Nx*gpu_id_y*(Ny/N_GPU_y);  // shift of sites for each gpu, necessary for spin copy

			#pragma omp master
			{
				hipEventCreate(&start);
				hipEventCreate(&stop);
				total_time = 0.0;
				hipEventRecord(start, 0);
			}
			hipDeviceEnablePeerAccess(gid[gid_P2P[cpu_thread_id][0]],0);
			hipDeviceEnablePeerAccess(gid[gid_P2P[cpu_thread_id][1]],0);
			hipDeviceEnablePeerAccess(gid[gid_P2P[cpu_thread_id][2]],0);
			hipDeviceEnablePeerAccess(gid[gid_P2P[cpu_thread_id][3]],0);
			#pragma omp master
			{
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&gpu_time, start, stop);
				total_time += gpu_time;
				printf("GPU time for enabling peer to peer is %.4f ms.\n", gpu_time);		
			}

			hipMalloc((void **)&d_spin_GPU[cpu_thread_id], size_site_GPU);
			
			#pragma omp master
			{
				hipEventRecord(start, 0);
			}
			// initialize the spin
			if (ini=='c')
				INITIALIZE_COLD<<<BpG, TpB>>> (d_spin_GPU[cpu_thread_id]);
			else
			{
				hipMalloc((void**)&devstate_GPU[cpu_thread_id], N_site/N_GPU*sizeof(hiprandState));
				SET_UP_KERNEL<<<N_block, N_thread>>> ( seed_GPU[cpu_thread_id], devstate_GPU[cpu_thread_id]);
				INITIALIZE_HOT<<<BpG, TpB>>> (d_spin_GPU[cpu_thread_id], devstate_GPU[cpu_thread_id]);
			}
			// spin initialize check
//			for (int i=0; i<Ny/N_GPU_y; i++)
//				hipMemcpy(h_spin+gid_P2P[cpu_thread_id][4]+i*Nx/N_GPU_x, d_spin_GPU[cpu_thread_id]+i*Nx/N_GPU_x, Nx/N_GPU_x*sizeof(int), hipMemcpyDeviceToHost);
//			#pragma omp barrier
//			#pragma omp single
//			{
//				FILE *check = fopen("initial_spin_check.txt", "w");
//				for (int j=0; j<Ny; j++)
//				{
//					for (int i=0; i<Nx; i++)	
//						fprintf(check, "%d\t", h_spin[i+j*Nx]);
//					fprintf(check, "\n");
//				}
//				fprintf(check, "\n");
//			}
				//
	
			if (mode=='b')
			{
				for (int i=0; i<Ny/N_GPU_y; i++)
					hipMemcpy(h_spin+gid_P2P[cpu_thread_id][4]+i*Nx/N_GPU_x, d_spin_GPU[cpu_thread_id]+i*Nx/N_GPU_x, Nx/N_GPU_x*sizeof(int), hipMemcpyDeviceToHost);
			}
			//
		
			#pragma omp master
			{
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&gpu_time, start, stop);
				total_time += gpu_time;
				printf("GPU time for initialize the spin lattice on device is %.4f ms.\n", gpu_time);		
			}
	
//			int block_o, block_e;
			int thread_o, thread_e;
//			block_o = N_block/2;
//			block_e = (N_block+1)/2;
//			block_e = block_o;
			thread_o = N_thread/2;
//			thread_e = (N_thread+1)/2;
			thread_e = thread_o;
//			int *pointer_of_threads;
			hipMalloc((void**)&odd_T_in_B_GPU[cpu_thread_id], thread_o*5*sizeof(int));
			hipMalloc((void**)&even_T_in_B_GPU[cpu_thread_id], thread_e*5*sizeof(int));

			#pragma omp master			
			{
				puts("Initialize the mapping from shared memory indices to thread indices.");
				hipEventRecord(start, 0);
			}
			DOUBLE_CHECKERBOARD_SETUP <<<1, TpB>>> (odd_T_in_B_GPU[cpu_thread_id], even_T_in_B_GPU[cpu_thread_id]);
			#pragma omp master
			{
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&gpu_time, start, stop);
				total_time += gpu_time;
				printf("GPU time for initializing the mapping from shared memory indices to thread indices is %.4f ms.\n", gpu_time);		
			}
			
			double **exp_update = (double **)malloc(2*sizeof(double *));
			exp_update[0] = (double *)malloc(5*sizeof(double));
			exp_update[1] = (double *)malloc(2*sizeof(double));
			#pragma omp master
			{
				puts("Calculate the Boltzmann factor and store it in the constant memroy.");
				hipEventRecord(start, 0);
			}
			UPDATE_MATRIX (exp_update[0], exp_update[1]);
//			for (int i=0; i<5; i++)
//				printf("%.4f\n", exp_update[0][i]);
//			for (int i=0; i<2; i++)
//				printf("%.4f\n", exp_update[1][i]);
			hipMemcpyToSymbol(HIP_SYMBOL(exp_update0_dev), exp_update[0], 5*sizeof(double));
			hipMemcpyToSymbol(HIP_SYMBOL(exp_update1_dev), exp_update[1], 2*sizeof(double));
			#pragma omp master
			{
				hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&gpu_time, start, stop);
				total_time += gpu_time;
				printf("GPU time for calculating the Boltzmann factor and storing it in the constant memory is %.4f ms.\n", gpu_time);		

				puts("index of Meas.\t<M>\t\t<E>\n===============================================================");
			}
//			hipFree(pointer_of_threads);
			free(exp_update);
			
			hipMalloc((void **)&d_M_GPU[cpu_thread_id], size_block);
			hipMalloc((void **)&d_E_GPU[cpu_thread_id], size_block);
//			hipMemset(d_M_GPU[cpu_thread_id], 0, size_block);
			hipMalloc((void**)&devstate_GPU[cpu_thread_id], N_site/N_GPU/2/2*sizeof(hiprandState));
			SET_UP_KERNEL<<<N_block/2, N_thread/2>>> (seed_GPU[cpu_thread_id], devstate_GPU[cpu_thread_id]);

			//constant memory test
//			hipEvent_t start_test, stop_test;
//			hipEventCreate(&start_test);
//			hipEventCreate(&stop_test);
//			hipEventRecord(start_test, 0);
//			CONSTANT_MEMORY_TEST<<<1,1>>> ();
//			hipEventRecord(stop_test, 0);
//			hipEventSynchronize(stop_test);
//			hipEventElapsedTime(&gpu_time, start_test, stop_test);
//			printf("GPU time for constant memory test is %.4f ms.\n", gpu_time);		
		  	//
		  	//hiprand device API seed test
//			hipMalloc((void**)&devstate_GPU[cpu_thread_id], sizeof(hiprandState));
//			SET_UP_KERNEL<<<N_block, N_thread>>> (seed, devstate_GPU[cpu_thread_id]);
//			for (int i=0; i<1; i++)
//				RANDOM_NUMBER_TEST<<<dim3(bx,by), dim3(tx,ty)>>> (i, devstate_GPU[cpu_thread_id]);
			//
			hipDeviceSynchronize();
			#pragma omp barrier
		}  // end of openMP

		// simulation
		M_mean = 0;
		E_mean = 0;
		hipEventRecord(start, 0);
			// thermalizatoin
		for (int N=0; N<N_thermal/interval_measure; N++)
		{
			#pragma omp parallel private (cpu_thread_id)
			{
				cpu_thread_id = omp_get_thread_num();

				ISING_ODD_BLOCK<<<dim3(bx/2,by), dim3(tx/2,ty), size_shared>>> (interval_measure, B, d_spin_GPU[cpu_thread_id], d_spin_GPU[gid_P2P[cpu_thread_id][0]], d_spin_GPU[gid_P2P[cpu_thread_id][1]], d_spin_GPU[gid_P2P[cpu_thread_id][2]], d_spin_GPU[gid_P2P[cpu_thread_id][3]], devstate_GPU[cpu_thread_id], odd_T_in_B_GPU[cpu_thread_id], even_T_in_B_GPU[cpu_thread_id]);
				hipDeviceSynchronize();
				#pragma omp barrier

				ISING_EVEN_BLOCK<<<dim3(bx/2,by), dim3(tx/2,ty), size_shared>>> (interval_measure, B, d_spin_GPU[cpu_thread_id], d_spin_GPU[gid_P2P[cpu_thread_id][0]], d_spin_GPU[gid_P2P[cpu_thread_id][1]], d_spin_GPU[gid_P2P[cpu_thread_id][2]], d_spin_GPU[gid_P2P[cpu_thread_id][3]], devstate_GPU[cpu_thread_id], odd_T_in_B_GPU[cpu_thread_id], even_T_in_B_GPU[cpu_thread_id]);
				hipDeviceSynchronize();
				#pragma omp barrier
			}	
		}
			//
			// measurements and updating
		for (int N=0; N<N_measure; N++)
		{
			#pragma omp parallel private(cpu_thread_id)
			{
				// measurement
				cpu_thread_id = omp_get_thread_num();
				ISING_MEASUREMENT<<<BpG, TpB, size_shared_measure>>> (B, d_spin_GPU[cpu_thread_id], d_spin_GPU[gid_P2P[cpu_thread_id][1]], d_spin_GPU[gid_P2P[cpu_thread_id][3]], d_M_GPU[cpu_thread_id], d_E_GPU[cpu_thread_id]);
				hipDeviceSynchronize();
				//
				hipMemcpy(h_M+N_block*cpu_thread_id, d_M_GPU[cpu_thread_id], size_block, hipMemcpyDeviceToHost);
				hipMemcpy(h_E+N_block*cpu_thread_id, d_E_GPU[cpu_thread_id], size_block, hipMemcpyDeviceToHost);
				M_GPU[cpu_thread_id] = 0;
				E_GPU[cpu_thread_id] = 0;
				for (int i=0; i<N_block; i++)
				{
					M_GPU[cpu_thread_id] += h_M[i+N_block*cpu_thread_id];
					E_GPU[cpu_thread_id] += h_E[i+N_block*cpu_thread_id];
				}			
  		}
			M_save[N] = 0;
			E_save[N] = 0;
			for (int i=0; i<N_GPU; i++)
			{
//				M_save[N] += M_GPU[i];
				M_save[N] += abs(M_GPU[i]);
				E_save[N] += E_GPU[i];
			}
			M_mean += M_save[N];
			E_mean += E_save[N];
				//
				//save
			if ((N+1)%to_save==0)
			{
				#pragma omp parallel private(cpu_thread_id)
				{
					cpu_thread_id = omp_get_thread_num();
					for (int i=0; i<Ny/N_GPU_y; i++)
						hipMemcpy(h_spin+gid_P2P[cpu_thread_id][4]+i*Nx/N_GPU_x, d_spin_GPU[cpu_thread_id]+i*Nx/N_GPU_x, Nx/N_GPU_x*sizeof(int), hipMemcpyDeviceToHost);
					#pragma omp barrier
				}
				fprintf(output, "#%d Spin Configuration\n===============================================================\n", (N+1)/to_save);
				for (int j=0; j<Ny; j++)
				{
					for (int i=0; i<Nx; i++)	
						fprintf(output, "%d\t", h_spin[i+j*Nx]);
					fprintf(output, "\n");
				}
				fprintf(output, "\n");
			}
				//
			if ((N+1)%to_display==0)
				printf("%d\t\t%.6f\t\t%.6f\n", N+1, M_mean/(N+1)/N_site, E_mean/(N+1)/N_site ); 
				//updating
			#pragma omp parallel private(cpu_thread_id)
			{
				cpu_thread_id = omp_get_thread_num();

				ISING_ODD_BLOCK<<<dim3(bx/2,by), dim3(tx/2,ty), size_shared>>> (interval_measure, B, d_spin_GPU[cpu_thread_id], d_spin_GPU[gid_P2P[cpu_thread_id][0]], d_spin_GPU[gid_P2P[cpu_thread_id][1]], d_spin_GPU[gid_P2P[cpu_thread_id][2]], d_spin_GPU[gid_P2P[cpu_thread_id][3]], devstate_GPU[cpu_thread_id], odd_T_in_B_GPU[cpu_thread_id], even_T_in_B_GPU[cpu_thread_id]);
				hipDeviceSynchronize();
				#pragma omp barrier
				ISING_EVEN_BLOCK<<<dim3(bx/2,by), dim3(tx/2,ty), size_shared>>> (interval_measure, B, d_spin_GPU[cpu_thread_id], d_spin_GPU[gid_P2P[cpu_thread_id][0]], d_spin_GPU[gid_P2P[cpu_thread_id][1]], d_spin_GPU[gid_P2P[cpu_thread_id][2]], d_spin_GPU[gid_P2P[cpu_thread_id][3]], devstate_GPU[cpu_thread_id], odd_T_in_B_GPU[cpu_thread_id], even_T_in_B_GPU[cpu_thread_id]);
				hipDeviceSynchronize();
				#pragma omp barrier
			}
		}  // end of measurement and updating

//		#pragma omp parallel private(cpu_thread_id)
//		{
//			cpu_thread_id = omp_get_thread_num();
//			ISING_MEASUREMENT<<<BpG, TpB, size_shared_measure>>> (B, d_spin_GPU[cpu_thread_id], d_spin_GPU[gid_P2P[cpu_thread_id][1]], d_spin_GPU[gid_P2P[cpu_thread_id][3]], d_M_GPU[cpu_thread_id], d_E_GPU[cpu_thread_id]);
//			hipDeviceSynchronize();
////			printf("%d\t%d\t%d\n", gid_P2P[cpu_thread_id][1], cpu_thread_id, gid_P2P[cpu_thread_id][3]);
//			hipMemcpy(h_M+N_block*cpu_thread_id, d_M_GPU[cpu_thread_id], size_block, hipMemcpyDeviceToHost);
//			hipMemcpy(h_E+N_block*cpu_thread_id, d_E_GPU[cpu_thread_id], size_block, hipMemcpyDeviceToHost);
//			M_GPU[cpu_thread_id] = 0;
//			E_GPU[cpu_thread_id] = 0;
//			for (int i=0; i<N_block; i++)
//			{
//				M_GPU[cpu_thread_id] += h_M[i+N_block*cpu_thread_id];
//				E_GPU[cpu_thread_id] += h_E[i+N_block*cpu_thread_id];
//			}			
//		}
//		double M_test = 0;
//		double E_test = 0;
//		for (int i=0; i<N_GPU; i++)
//		{
//			M_test += abs(M_GPU[i]);
//			E_test += E_GPU[i];
//		}
//		printf("%.6f\t%.6f\n", M_test/N_site, E_test/N_site);

			//analysis
		puts("===============================================================");
		fprintf(output2, "<M>\t<E>\n===============================================================\n");
		M_mean /= (double)(N_site)*(double)(N_measure);
		E_mean /= (double)(N_site)*(double)(N_measure);
		for (int i=0; i<N_measure; i++)
		{
//			M_sigma += pow(M_save[i]/N_site-M_mean, 2.0);
			M_sigma += pow(abs(M_save[i])/N_site-M_mean, 2.0);
			E_sigma += pow(E_save[i]/N_site-E_mean, 2.0);
			fprintf(output2, "%.6f\t%.6f\n", M_save[i]/N_site, E_save[i]/N_site);
		}
		M_sigma = sqrt(M_sigma/(N_measure-1));
		E_sigma = sqrt(E_sigma/(N_measure-1));
		puts("GPU Simulation result:");
		printf("\tMagnetization: Mean = %.6f\t Stand Deviation = %.6f\n", M_mean, M_sigma);
		printf("\tEnergy: Mean = %.6f\t Stand Deviation = %.6f\n", E_mean, E_sigma);
		// measurement and updating end
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		total_time += gpu_time;
		printf("GPU time for simulation is %.4f ms.\n", gpu_time);		
		printf("Total GPU time is %.4f ms.\n", total_time);		
			//
		//  simulation ends

		#pragma omp parallel private(cpu_thread_id)
		{
			cpu_thread_id = omp_get_thread_num();
			hipFree(d_spin_GPU[cpu_thread_id]);
			hipFree(d_M_GPU[cpu_thread_id]);
			hipFree(d_E_GPU[cpu_thread_id]);
			hipFree(devstate_GPU[cpu_thread_id]);
			hipDeviceDisablePeerAccess(gid[gid_P2P[cpu_thread_id][0]]);
			hipDeviceDisablePeerAccess(gid[gid_P2P[cpu_thread_id][1]]);
			hipDeviceDisablePeerAccess(gid[gid_P2P[cpu_thread_id][2]]);
			hipDeviceDisablePeerAccess(gid[gid_P2P[cpu_thread_id][3]]);
			free(gid_P2P[cpu_thread_id]);
			if (cpu_thread_id!=0)
				hipDeviceReset();
		}
		free(odd_T_in_B_GPU);
		free(even_T_in_B_GPU);
		free(devstate_GPU);
		free(d_spin_GPU);
		free(d_M_GPU);
		free(d_E_GPU);
		free(h_M);
		free(h_E);
		free(M_GPU);
		free(E_GPU);
		free(M_save);
		free(E_save);
		free(gid_P2P);
		fclose(output);
		fclose(output2);
	}

	if (mode!='g')
	{
		rng = gsl_rng_alloc(gsl_rng_mt19937);
		gsl_rng_set(rng, seed);
		if (mode=='c')
		{
			hipEventCreate(&start);
			hipEventCreate(&stop);
		}
		// initialize the spin (for mode 'b', the spin is already initialized by GPU)
		if (ini=='c')
			for (int i=0; i<N_site; i++)
				h_spin[i] = 1;
		else
			for (int i=0; i<N_site; i++)
			{
				if (gsl_rng_uniform(rng)<=0.5)
					h_spin[i] = 1;
				else
					h_spin[i] = -1;
			}
		//
		hipEventRecord(start, 0);
		ISING_CPU();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&cpu_time, start, stop);
		printf("CPU time for simulation is %.4f ms.\n", cpu_time);		
		if (mode=='b')
			printf("Speed-up by GPU is: %.2f\n", cpu_time/total_time);
		hipEventDestroy(stop);
	}
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceReset();
	free(h_spin);

	if ( B==0.0 )
	{
		EXACT_EM(&E_exact, &M_exact);
		printf("Exact solution: Tc = %.6f , M_exact = %.6f, E_exact = %.6f .\n", Tc, M_exact, E_exact);
	}
	else
		puts("Exact solution when B is non-zero is waiting to be found!");
}

__device__ int UPDATE_SPIN_GPU(int old_spin, int new_spin, int spin_around, double x)
{
	if (old_spin==1)
	{
		if (spin_around==4)
		{
			if (x<exp_update0_dev[0])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==2)
		{
			if (x<exp_update0_dev[1])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==0)
		{
			if (x<exp_update0_dev[2])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==-2)
		{
			if (x<exp_update0_dev[3])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==-4)
		{
			if (x<exp_update0_dev[4])
				return new_spin;
			else
				return old_spin;
		}
		else
		{
			printf("Error happens for old_spin parallel to B! Exit!\n");
			printf("Spin aournd = %d\n", spin_around);
			return 0;
		}
	}
	else if (old_spin==-1)
	{
		if (spin_around==-4)
		{
			if (x<exp_update1_dev[0])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==-2)
		{
			if (x<exp_update1_dev[1])
				return new_spin;
			else
				return old_spin;
		}
		else
		{
			printf("Error happens for old spin anti-parallel to B! Exit!\n");
			printf("Spin aournd = %d\n", spin_around);
			return 0;
		}
	}
	else
	{
		printf("Error for old spin! Exit!\n");
		printf("Old spin = %d\n", old_spin);
		return 0;
	}
}

void EXACT_EM (double *E_exact, double *M_exact)
{
	double k = 1.0/sinh(2.0/T)/sinh(2.0/T);
	Tc =  2.0/log(1.0+sqrt(2.0));

	*E_exact = -1.0/tanh(2.0/T)*( 1.0 + 2.0/M_PI*(2.0*tanh(2.0/T)*tanh(2.0/T)-1.0)*gsl_sf_ellint_Kcomp(2.0*sqrt(k)/(1.0+k), GSL_PREC_DOUBLE) );

	if ( T<=Tc )
		*M_exact = pow( (1.0-pow(sinh(2.0/T), -4.0)), 0.125);
	else
		*M_exact = 0.0;
}

void UPDATE_MATRIX (double *A1, double *A2)
{
	A1[0] = exp(-2.0*(4.0+B)/T);
	A1[1] = exp(-2.0*(2.0+B)/T);
	A1[2] = exp(-2.0*B/T);
	A1[3] = exp(-2.0*(-2.0+B)/T);
	A1[4] = exp(-2.0*(-4.0+B)/T);

	A2[0] = exp(-2.0*(4.0-B)/T);
	A2[1] = exp(-2.0*(2.0-B)/T);
}

//void UPDATE_SPIN(int center, int old_spin, int new_spin, int spin_around, double delta_E, int *h_spin, double **exp_update, gsl_rng *rng)
void UPDATE_SPIN(int center, int old_spin, int new_spin, int spin_around, int delta_E, double **exp_update)
{
	if (delta_E<=0)
		h_spin[center] = new_spin;
	else if (delta_E>0)
	{
		if (old_spin==1)
		{
			if (spin_around==4)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][0])
					h_spin[center] = new_spin;
			}
			else if (spin_around==2)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][1])
					h_spin[center] = new_spin;
			}
			else if (spin_around==0)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][2])
					h_spin[center] = new_spin;
			}
			else if (spin_around==-2)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][3])
					h_spin[center] = new_spin;
			}
			else if (spin_around==-4)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][4])
					h_spin[center] = new_spin;
			}
			else
			{
				puts("Error happens for old_spin parallel to B! Exit!");
				printf("Spin aournd = %d\n", spin_around);
				exit(1);
			}
		}
		else if (old_spin==-1)
		{
			if (spin_around==-4)
			{
				if (gsl_rng_uniform(rng)<exp_update[1][0])
					h_spin[center] = new_spin;
			}
			else if (spin_around==-2)
			{
				if (gsl_rng_uniform(rng)<exp_update[1][1])
					h_spin[center] = new_spin;
			}
			else
			{
				puts("Error happens for old spin anti-parallel to B! Exit!");
				printf("Spin aournd = %d\n", spin_around);
				exit(1);
			}
		}
		else
		{
			puts("Error for old spin! Exit!");
			printf("Old spin = %d\n", old_spin);
			exit(1);
		}
	} // end of delta_E>0 case
}

void ISING_CPU (void)
{
	int x, y, parity;
	int center, fw_x, bw_x, fw_y, bw_y, spin_around;
	int old_spin, new_spin, delta_E;
	int i_o = (int)(N_site/2);
	int i_e = (int)((N_site+1)/2);
	int *h_spin_odd = (int*)malloc(i_o*sizeof(int));
	int *h_spin_even = (int*)malloc(i_e*sizeof(int));
	double **exp_update = (double**)malloc(2*sizeof(double*));
	exp_update[0] = (double*)malloc(5*sizeof(double));
	exp_update[1] = (double*)malloc(2*sizeof(double));
//	gsl_rng rng;
//	rng = gsl_rng_alloc(gsl_rng_mt19937);
//	gsl_rng_set(rng, seed);
	UPDATE_MATRIX(exp_update[0], exp_update[1]);
	
	//odd site
	for (int i=0; i<i_o; i++)
	{
		x = (2*i)%Nx;
		y = (2*i)/Nx;
		parity = (x+y+1)%2;
		x = x+parity;
		h_spin_odd[i] = x+Nx*y;
	}
	//
	//even site
	for (int i=0; i<i_e; i++)
	{
		x = (2*i)%Nx;
		y = (2*i)/Nx;
		parity = (x+y)%2;
		x = x+parity;
		h_spin_even[i] = x+Nx*y;
	}
	//
	for (int N=0; N<N_thermal; N++)
	{
		for (int i=0; i<i_o; i++)
		{
			center = h_spin_odd[i];
			x = center%Nx;
			y = center/Nx;
			fw_x = (x+1)%Nx+y*Nx;
			bw_x = (x-1+Nx)%Nx+y*Nx;
			fw_y = x+((y+1)%Ny)*Nx;
			bw_y = x+((y-1+Ny)%Ny)*Nx;
			spin_around = h_spin[fw_x]+h_spin[bw_x]+h_spin[fw_y]+h_spin[bw_y];
			old_spin = h_spin[center];
			new_spin = -old_spin;
			delta_E = (old_spin-new_spin)*(spin_around+B); // delta_E = E_new-E_ole = -(spin_new[center]-spin_old[center])*(summation_of_neighboring_spin+B)
//			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, h_spin, exp_update, rng);
			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, exp_update);
		} // end of odd site
		for (int i=0; i<i_e; i++)
		{
			center = h_spin_even[i];
			x = center%Nx;
			y = center/Nx;
			fw_x = (x+1)%Nx+y*Nx;
			bw_x = (x-1+Nx)%Nx+y*Nx;
			fw_y = x+((y+1)%Ny)*Nx;
			bw_y = x+((y-1+Ny)%Ny)*Nx;
			spin_around = h_spin[fw_x]+h_spin[bw_x]+h_spin[fw_y]+h_spin[bw_y];
			old_spin = h_spin[center];
			new_spin = -old_spin;
			delta_E = (old_spin-new_spin)*(spin_around+B); // delta_E = E_new-E_ole = -(spin_new[center]-spin_old[center])*(summation_of_neighboring_spin+B)
//			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, h_spin, exp_update, rng);
			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, exp_update);
		} // end even_site
	} // end of thermalization

	int N = 0;
	int count = 1;
	double M_mean = 0, E_mean = 0;
	double M_sigma = 0, E_sigma = 0;
	double *M_save = (double*)malloc(N_measure*sizeof(double));
	double *E_save = (double*)malloc(N_measure*sizeof(double));
	FILE *output = fopen("spin_config_cpu.txt", "w");
	FILE *output2 = fopen("M_and_E_cpu.txt", "w");
	puts("index of Meas.\t<M>\t\t<E>\n===============================================================");
	while (N<N_measure)
	{
		if (count%interval_measure==0)
		{
			int M = 0, E = 0;
			for (int i=0; i<N_site; i++)
			{
				x = i%Nx;
				y = i/Nx;
				fw_x = (x+1)%Nx+y*Nx;
				fw_y = x+((y+1)%Ny)*Nx;
				spin_around = h_spin[fw_x]+h_spin[fw_y];
				M += h_spin[i];
				E += -h_spin[i]*(spin_around+B);
			}
//			M_mean += M;
			M_mean += abs(M);
			E_mean += E;
			M_save[N] = M;
			E_save[N] = E;
			N++;
		}

		if (count%interval_save==0)
		{
			fprintf(output, "#%d Spin Configuration\n===============================================================\n", (int)(count/interval_save));
			for (int j=0; j<Ny; j++)
			{
				for (int i=0; i<Nx; i++)	
					fprintf(output, "%d\t", h_spin[i+j*Nx]);
				fprintf(output, "\n");
			}
			fprintf(output, "\n");
		}
		
		if (count%interval_display==0)
			printf("%d\t\t%.6f\t\t%.6f\n", N, M_mean/N/N_site, E_mean/N/N_site ); 

		for (int i=0; i<i_o; i++)
		{
			center = h_spin_odd[i];
			x = center%Nx;
			y = center/Nx;
			fw_x = (x+1)%Nx+y*Nx;
			bw_x = (x-1+Nx)%Nx+y*Nx;
			fw_y = x+((y+1)%Ny)*Nx;
			bw_y = x+((y-1+Ny)%Ny)*Nx;
			spin_around = h_spin[fw_x]+h_spin[bw_x]+h_spin[fw_y]+h_spin[bw_y];
			old_spin = h_spin[center];
			new_spin = -old_spin;
			delta_E = (old_spin-new_spin)*(spin_around+B); // delta_E = E_new-E_ole = -(spin_new[center]-spin_old[center])*(summation_of_neighboring_spin+B)
//			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, h_spin, exp_update, rng);
			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, exp_update);
		} // end of odd site
		for (int i=0; i<i_e; i++)
		{
			center = h_spin_even[i];
			x = center%Nx;
			y = center/Nx;
			fw_x = (x+1)%Nx+y*Nx;
			bw_x = (x-1+Nx)%Nx+y*Nx;
			fw_y = x+((y+1)%Ny)*Nx;
			bw_y = x+((y-1+Ny)%Ny)*Nx;
			spin_around = h_spin[fw_x]+h_spin[bw_x]+h_spin[fw_y]+h_spin[bw_y];
			old_spin = h_spin[center];
			new_spin = -old_spin;
			delta_E = (old_spin-new_spin)*(spin_around+B); // delta_E = E_new-E_ole = -(spin_new[center]-spin_old[center])*(summation_of_neighboring_spin+B)
//			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, h_spin, exp_update, rng);
			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, exp_update);
		} // end even_site
		count++;
	} // end of measurements
	puts("===============================================================");
	fprintf(output2, "<M>\t<E>\n===============================================================\n");
	M_mean /= (N_site*N_measure);
	E_mean /= (N_site*N_measure);
	for (int i=0; i<N_measure; i++)
	{
//		M_sigma += pow(M_save[i]/N_site-M_mean, 2.0);
		M_sigma += pow(abs(M_save[i])/N_site-M_mean, 2.0);
		E_sigma += pow(E_save[i]/N_site-E_mean, 2.0);
		fprintf(output2, "%.6f\t%.6f\n", M_save[i]/N_site, E_save[i]/N_site);
	}
	M_sigma = sqrt(M_sigma/(N_measure-1));
	E_sigma = sqrt(E_sigma/(N_measure-1));
	puts("CPU Simulation result:");
	printf("\tMagnetization: Mean = %.6f\t Stand Deviation = %.6f\n", M_mean, M_sigma);
	printf("\tEnergy: Mean = %.6f\t Stand Deviation = %.6f\n", E_mean, E_sigma);

	free(h_spin_odd);
	free(h_spin_even);
	free(exp_update);
	free(M_save);
	free(E_save);
	fclose(output);
	fclose(output2);
} // end of ISING_CPU


