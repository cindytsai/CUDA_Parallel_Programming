#include "hip/hip_runtime.h"
# include <stdio.h>
# include <stdlib.h>
# include <string.h>
# include <math.h>
# include <omp.h>
# include <hip/hip_runtime.h>
# include <hiprand/hiprand_kernel.h>
# include <gsl/gsl_sf_ellint.h>
# include <gsl/gsl_rng.h>
# define _USE_MATH_DEFINES

__constant__ double exp_update0_dev[5];
__constant__ double exp_update1_dev[2];
__device__ int UPDATE_SPIN_GPU (int, int, int, double);

void EXACT_EM (double*, double*);
void UPDATE_MATRIX(double*, double*);
void UPDATE_SPIN(int, int, int, int, int, int*, double*, gsl_rng*);
void ISING_CPU (void);

__global__ void SET_UP_KERNEL (long seed, hiprandState *state)
{
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	hiprand_init (seed, index, 0, &state[index]);
}

__global__ void INITIALIZE_COLD (int *d_spin)
{
	int index_x = threadIdx.x + blockIdx.x*blockDim.x;
	int index_y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = index_x + index_y*blockDim.x*gridDim.x;

	d_spin[index] = 1;
}

__global__ void INITIALIZE_HOT (int *d_spin, hiprandState *state)
{
	int index_x = threadIdx.x + blockIdx.x*blockDim.x;
	int index_y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = index_x + index_y*blockDim.x*gridDim.x;
	hiprandState local = state[index];
	double x = hiprand_uniform(&local);

	if (x<0.5)
		d_spin[index] = 1;
	else
		d_spin[index] = -1;
//	printf("%d\t%.4f\n", index, x);
	state[index] = local;
}

__global__ void DOUBLE_CHECKERBOARD_SETUP(int *oTiB, int *eTiB)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Tx = blockDim.x;
	int thread_index = tx + ty*Tx;
	int cache_index = (tx+1) + (ty+1)*(Tx+2);
	int cache_index_r = (tx+2) + (ty+1)*(Tx+2);
	int cache_index_l = tx + (ty+1)*(Tx+2);
	int cache_index_u = (tx+1) + (ty+2)*(Tx+2);
	int cache_index_d = (tx+1) + ty*(Tx+2);
	// odd threads
	if (threadIdx.x%2!=threadIdx.y%2)
	{
		oTiB[5*(thread_index/2)] = cache_index;
		oTiB[5*(thread_index/2)+1] = cache_index_r;
		oTiB[5*(thread_index/2)+2] = cache_index_l;
		oTiB[5*(thread_index/2)+3] = cache_index_u;
		oTiB[5*(thread_index/2)+4] = cache_index_d;
	}
	//
	// even threads
	else if (threadIdx.x%2==threadIdx.y%2)
	{
		eTiB[5*(thread_index/2)] = cache_index;
		eTiB[5*(thread_index/2)+1] = cache_index_r;
		eTiB[5*(thread_index/2)+2] = cache_index_l;
		eTiB[5*(thread_index/2)+3] = cache_index_u;
		eTiB[5*(thread_index/2)+4] = cache_index_d;
//			printf("%d\t%d\t%d\t\n", block_index, cache_index_d, eTioB[block_index/2][5*(thread_index/2)+4]);
	}
	//
}

__global__ void ISING_ODD_BLOCK(int interval_measure, double B, int *d_spin, hiprandState *state, int *oTiB, int *eTiB)
{
	extern __shared__ int cache[];
	int lattice_center_odd, lattice_center_even;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Tx = blockDim.x;
	int Ty = blockDim.y;
	int thread_index = tx + ty*Tx;
	int block_index = blockIdx.x + gridDim.x*blockIdx.y;
	int Bx = 2*gridDim.x;
	int By = gridDim.y;
	int Nx = 2*Tx*Bx;
	int Ny = Ty*By;
	int bx = (2*block_index)%Bx;
	int by = (2*block_index)/Bx;
	//for odd block
	int parity = (bx+by+1)%2;
	bx += parity;
	// mount odd site in each odd block to shared
	int x = (2*thread_index)%(2*Tx);
	int y = (2*thread_index)/(2*Tx);
	parity = (x+y+1)%2;
	x += parity;
	int cache_index = (x+1) + (y+1)*(2*Tx+2);
	x += 2*Tx*bx;
	y += Ty*by;
	int lattice_index = x + y*Nx;
	lattice_center_odd = lattice_index;  // to copy the updated spin from shared to d_spin
	cache[cache_index] = d_spin[lattice_index];
//	if (block_index==0)
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
	//
	// mount even site in each block to shared
	x = (2*thread_index)%(2*Tx);
	y = (2*thread_index)/(2*Tx);
	parity = (x+y)%2;
	x += parity;
	cache_index = (x+1) + (y+1)*(2*Tx+2);
	x += 2*Tx*bx;
	y += Ty*by;
	lattice_index = x + y*Nx;
	lattice_center_even = lattice_index;  // to copy the updated spin from shared to d_spin
	cache[cache_index] = d_spin[lattice_index];
//	if (block_index==2047)
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
	//
	// mount the boundary to shared
	if (Tx==1)
	{
		// left
		x = 0;
		y = ty; 
		cache_index = (y+1)*(2*Tx+2);
		x += 2*Tx*bx;
		y += Ty*by;
		lattice_index = (x-1+Nx)%Nx + y*Nx;
		cache[cache_index] = d_spin[lattice_index];
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		// right
		x = 1;
		y = ty;
		cache_index = (x+2) + (y+1)*(2*Tx+2);
		x += 2*Tx*bx;
		y += Ty*by;
		lattice_index = (x+1)%Nx + y*Nx;
		cache[cache_index] = d_spin[lattice_index];
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		// down
		if (ty==0)
		{
			for (int i=0; i<2; i++)
			{
				x = i;
				y = 0;
				cache_index = x+1;
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y-1+Ny)%Ny*Nx;
				cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
		// up
		else if (ty==Ty-1)
		{
			for (int i=0; i<2; i++)
			{
				x = i;
				y = Ty-1;
				cache_index = (x+1) + (y+2)*(2*Tx+2);
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y+1)%Ny*Nx;
				cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
	}
	else
	{
		// left
		if (tx==0)
		{
			x = 0;
			y = ty;
			cache_index = (y+1)*(2*Tx+2);
			x += 2*Tx*bx;
			y += Ty*by;
			lattice_index = (x-1+Nx)%Nx + y*Nx;
			cache[cache_index] = d_spin[lattice_index];
//			printf("%d\t%d\n", bx*Bx, by*By);
//			printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		}
		// right
		else if (tx==Tx-1)
		{
			x = 2*Tx-1;
			y = ty;
			cache_index = (x+2) + (y+1)*(2*Tx+2);
			x += 2*Tx*bx;
			y += Ty*by;
			lattice_index = (x+1)%Nx + y*Nx;
			cache[cache_index] = d_spin[lattice_index];
//			printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		}
		// down
		if (ty==0)
		{
			for (int i=0; i<2; i++)
			{
				x = 2*tx+i;
				y = 0;
				cache_index = x+1;
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y-1+Ny)%Ny*Nx;
				cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
		// up
		else if (ty==Ty-1)
		{
			for (int i=0; i<2; i++)
			{
				x = 2*tx+i;
				y = Ty-1;
				cache_index = (x+1) + (y+2)*(2*Tx+2);
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y+1)%Ny*Nx;
				cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
	}
	__syncthreads();
	//
	hiprandState local = state[threadIdx.x+blockDim.x*blockIdx.x + blockDim.x*gridDim.x*(threadIdx.y+blockDim.y*blockIdx.y)];
	int center, right, left, up, down;
	int spin_r, spin_l, spin_u, spin_d;
	int old_spin, new_spin, spin_around;
	int delta_E;
	// spin update
		// odd threads
	center = oTiB[5*thread_index];
	right = oTiB[5*thread_index+1];
	left = oTiB[5*thread_index+2];
	up = oTiB[5*thread_index+3];
	down = oTiB[5*thread_index+4];

	for (int iter=0; iter<interval_measure; iter++)
	{
		spin_r = cache[right];
		spin_l = cache[left];
		spin_u = cache[up];
		spin_d = cache[down];
		old_spin = cache[center];
		new_spin = -old_spin;
		spin_around = spin_r + spin_l + spin_u  + spin_d;
		delta_E = (old_spin-new_spin)*(spin_around+B);
		if (delta_E<=0)
			cache[center] = new_spin;
		else
		{
			double x = hiprand_uniform(&local);
			cache[center] = UPDATE_SPIN_GPU(old_spin, new_spin, spin_around, x);
//			if (thread_index==0)
//				printf("%.4f\n", x);
		}
	}
	d_spin[lattice_center_odd] = cache[center];
	__syncthreads();
		//
		// even threads;
	center = eTiB[5*thread_index];
	right = eTiB[5*thread_index+1];
	left = eTiB[5*thread_index+2];
	up = eTiB[5*thread_index+3];
	down = eTiB[5*thread_index+4];

	for (int iter=0; iter<interval_measure; iter++)
	{
		spin_r = cache[right];
		spin_l = cache[left];
		spin_u = cache[up];
		spin_d = cache[down];
		old_spin = cache[center];
		new_spin = -old_spin;
		spin_around = spin_r + spin_l + spin_u  + spin_d;
		delta_E = (old_spin-new_spin)*(spin_around+B);
		if (delta_E<=0)
			cache[center] = new_spin;
		else
		{
			double x = hiprand_uniform(&local);
			cache[center] = UPDATE_SPIN_GPU(old_spin, new_spin, spin_around, x);
		}
	}
	d_spin[lattice_center_even] = cache[center];
	state[threadIdx.x+blockDim.x*blockIdx.x + blockDim.x*gridDim.x*(threadIdx.y+blockDim.y*blockIdx.y)]=local;
	__syncthreads();
		//
}

__global__ void ISING_EVEN_BLOCK(int interval_measure, double B, int *d_spin, hiprandState *state, int *oTiB, int *eTiB)
{
	extern __shared__ int cache[];
	int lattice_center_odd, lattice_center_even;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int Tx = blockDim.x;
	int Ty = blockDim.y;
	int thread_index = tx + ty*Tx;
	int block_index = blockIdx.x + gridDim.x*blockIdx.y;
	int Bx = 2*gridDim.x;
	int By = gridDim.y;
	int Nx = 2*Tx*Bx;
	int Ny = Ty*By;
	int bx = (2*block_index)%Bx;
	int by = (2*block_index)/Bx;
	// for even block
	int parity = (bx+by)%2;
	bx += parity;
	// mount odd site in each odd block to shared
	int x = (2*thread_index)%(2*Tx);
	int y = (2*thread_index)/(2*Tx);
	parity = (x+y+1)%2;
	x += parity;
	int cache_index = (x+1) + (y+1)*(2*Tx+2);
	x += 2*Tx*bx;
	y += Ty*by;
	int lattice_index = x + y*Nx;
	lattice_center_odd = lattice_index; // to copy the updated spin from shared to d_spin
	cache[cache_index] = d_spin[lattice_index];
//	if (block_index==3)
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
	//
	// mount even site in each block to shared
	x = (2*thread_index)%(2*Tx);
	y = (2*thread_index)/(2*Tx);
	parity = (x+y)%2;
	x += parity;
	cache_index = (x+1) + (y+1)*(2*Tx+2);
	x += 2*Tx*bx;
	y += Ty*by;
	lattice_index = x + y*Nx;
	lattice_center_even = lattice_index; // to copy the updated spin from shared to d_spin
	cache[cache_index] = d_spin[lattice_index];
//	if (block_index==2)
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
	//
	// mount the boundary to shared
	if (Tx==1)
	{
		// left
		x = 0;
		y = ty; 
		cache_index = (y+1)*(2*Tx+2);
		x += 2*Tx*bx;
		y += Ty*by;
		lattice_index = (x-1+Nx)%Nx + y*Nx;
		cache[cache_index] = d_spin[lattice_index];
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		// right
		x = 1;
		y = ty;
		cache_index = (x+2) + (y+1)*(2*Tx+2);
		x += 2*Tx*bx;
		y += Ty*by;
		lattice_index = (x+1)%Nx + y*Nx;
		cache[cache_index] = d_spin[lattice_index];
//		printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		// down
		if (ty==0)
		{
			for (int i=0; i<2; i++)
			{
				x = i;
				y = 0;
				cache_index = x+1;
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y-1+Ny)%Ny*Nx;
				cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
		// up
		else if (ty==Ty-1)
		{
			for (int i=0; i<2; i++)
			{
				x = i;
				y = Ty-1;
				cache_index = (x+1) + (y+2)*(2*Tx+2);
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y+1)%Ny*Nx;
				cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
	}
	else
	{
		// left
		if (tx==0)
		{
			x = 0;
			y = ty;
			cache_index = (y+1)*(2*Tx+2);
			x += 2*Tx*bx;
			y += Ty*by;
			lattice_index = (x-1+Nx)%Nx + y*Nx;
			cache[cache_index] = d_spin[lattice_index];
//			printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		}
		// right
		else if (tx==Tx-1)
		{
			x = 2*Tx-1;
			y = ty;
			cache_index = (x+2) + (y+1)*(2*Tx+2);
			x += 2*Tx*bx;
			y += Ty*by;
			lattice_index = (x+1)%Nx + y*Nx;
			cache[cache_index] = d_spin[lattice_index];
//			printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
		}
		// down
		if (ty==0)
		{
			for (int i=0; i<2; i++)
			{
				x = 2*tx+i;
				y = 0;
				cache_index = x+1;
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y-1+Ny)%Ny*Nx;
				cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
		// up
		else if (ty==Ty-1)
		{
			for (int i=0; i<2; i++)
			{
				x = 2*tx+i;
				y = Ty-1;
				cache_index = (x+1) + (y+2)*(2*Tx+2);
				x += 2*Tx*bx;
				y += Ty*by;
				lattice_index = x + (y+1)%Ny*Nx;
				cache[cache_index] = d_spin[lattice_index];
//				printf("%d\t%d\t%d\t%d\n", lattice_index, cache_index, d_spin[lattice_index], cache[cache_index]);
			}
		}
	}
	__syncthreads();
	//
	hiprandState local = state[threadIdx.x+blockDim.x*blockIdx.x + blockDim.x*gridDim.x*(threadIdx.y+blockDim.y*blockIdx.y)];
	int center, right, left, up, down;
	int spin_r, spin_l, spin_u, spin_d;
	int old_spin, new_spin, spin_around;
	int delta_E;
	// spin update
		// odd threads
	center = oTiB[5*thread_index];
	right = oTiB[5*thread_index+1];
	left = oTiB[5*thread_index+2];
	up = oTiB[5*thread_index+3];
	down = oTiB[5*thread_index+4];

	for (int iter=0; iter<interval_measure; iter++)
	{
		spin_r = cache[right];
		spin_l = cache[left];
		spin_u = cache[up];
		spin_d = cache[down];
		old_spin = cache[center];
		new_spin = -old_spin;
		spin_around = spin_r + spin_l + spin_u  + spin_d;
		delta_E = (old_spin-new_spin)*(spin_around+B);
		if (delta_E<=0)
			cache[center] = new_spin;
		else
		{
			double x = hiprand_uniform(&local);
			cache[center] = UPDATE_SPIN_GPU(old_spin, new_spin, spin_around, x);
		}
	}
	d_spin[lattice_center_odd] = cache[center];
	__syncthreads();
		//
		// even threads;
	center = eTiB[5*thread_index];
	right = eTiB[5*thread_index+1];
	left = eTiB[5*thread_index+2];
	up = eTiB[5*thread_index+3];
	down = eTiB[5*thread_index+4];

	for (int iter=0; iter<interval_measure; iter++)
	{
		spin_r = cache[right];
		spin_l = cache[left];
		spin_u = cache[up];
		spin_d = cache[down];
		old_spin = cache[center];
		new_spin = -old_spin;
		spin_around = spin_r + spin_l + spin_u  + spin_d;
		delta_E = (old_spin-new_spin)*(spin_around+B);
		if (delta_E<=0)
			cache[center] = new_spin;
		else
		{
			double x = hiprand_uniform(&local);
			cache[center] = UPDATE_SPIN_GPU(old_spin, new_spin, spin_around, x);
		}
	}
	d_spin[lattice_center_even] = cache[center];
	state[threadIdx.x+blockDim.x*blockIdx.x + blockDim.x*gridDim.x*(threadIdx.y+blockDim.y*blockIdx.y)]=local;
	__syncthreads();
		//
}

__global__ void  ISING_MEASUREMENT (double B, int *d_spin, int *d_M, int *d_E)
{
	extern __shared__ int cache[];
	int layer = blockDim.x*blockDim.y/2;
	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	int Nx = blockDim.x*gridDim.x;
	int Ny = blockDim.y*gridDim.y;
	int index = x + Nx*y;
	int index_r = (x+1)%Nx + Nx*y;
	int index_u = x + Nx*((y+1)%Ny);
	int cache_index = threadIdx.x + blockDim.x*threadIdx.y;
	int block_index = blockIdx.x + gridDim.x*blockIdx.y;
	int shift = blockDim.x*blockDim.y;
	cache[cache_index] = d_spin[index];
	cache[cache_index+shift] = -d_spin[index]*(d_spin[index_r]+d_spin[index_u]+B);
	__syncthreads();

	while (layer>0)
	{
		if (cache_index<layer)
		{
			cache[cache_index] += cache[cache_index+layer];
			cache[cache_index+shift] += cache[cache_index+shift+layer];
		}
		layer /= 2;
		__syncthreads();
	}	
	if (cache_index==0)
	{
		d_M[block_index] = cache[0];
		d_E[block_index] = cache[shift];
	}
}

__global__ void CONSTANT_MEMORY_TEST ()
{
//	printf("Do constant memory test.\n");
	for (int i=0; i<5; i++)
		printf("%.4f\t", exp_update0_dev[i]);
	printf("\n");
	for (int i=0; i<2; i++)
		printf("%.4f\t", exp_update1_dev[i]);
	printf("\n");
}

__global__ void RANDOM_NUMBER_TEST (int iter, hiprandState *state)
{
	int index_x = threadIdx.x + blockIdx.x*blockDim.x;
	int index_y = threadIdx.y + blockIdx.y*blockDim.y;
	int index = index_x + index_y*blockDim.x*gridDim.x;
	hiprandState local = state[index];
	
	int N = 1;
	for (int i=0; i<N; i++)
		printf("%d\t%.4f\n", iter, hiprand_uniform(&local));
	state[threadIdx.x+blockDim.x*blockIdx.x] = local;
}

char ini;
int N_thermal, N_measure, interval_measure, interval_save, interval_display;
int Nx, Ny, N_site;
double T, B;
double Tc, E_exact, M_exact;
int *h_spin;
long seed;
gsl_rng *rng;

int main(void)
{
	char mode;
	int size_site, m, n;
	int gid, tx, ty, bx, by, N_block, N_thread;
	float gpu_time, total_time, cpu_time;
	hipEvent_t start, stop;
	
	puts("Simulate the 2D Ising model by one GPU.\n");
	puts("Set the lattice size ( Nx,Ny ) for the 2D lattice (Nx and Ny must be even to apply checkboard scheme with periodic boundary condition).");
	scanf("%d %d", &Nx, &Ny);
	N_site = Nx*Ny;
	if (Nx%2!=0||Ny%2!=0)
	{
		puts("Nx and Ny must be even! Exit!");
		exit(1);
	}
	printf("The lattice size is ( %d,%d ) .\n", Nx, Ny);

	size_site = N_site*sizeof(int);	
	h_spin = (int*)malloc(size_site);

	puts("Set the seed for random number generator.");
	scanf("%ld", &seed);
	printf("Seed is set to be %ld .\n", seed);
	puts("Set the temperature for the simulation in unit of J.");
	scanf("%lf", &T);
	if (T<0)
	{
		puts("Temperature must be positive! Exit!");
		exit(1);
	}
	printf("The temperature is %.6f J .\n", T);
	puts("Set the magnetic field for the simulation in unit of J.");
	scanf("%lf", &B);
	if (B<0)
	{
		puts("Field strength must be positive! Exit!");
		exit(1);
	}
	printf("The magnetic field is %.6f J.\n", B);
	puts("Choose cold/hot start (c/h) .");
	scanf("%c", &ini);  // absorb the \n
	scanf("%c", &ini);
	if ( ini=='c' )
		puts("Cold start is chosen.");
	else if ( ini=='h' )
		puts("Hot start is chosen");
	else
	{
		puts("Must enter c or h! Exit!");
		exit(1);
	}
	printf("\n");
	puts("Enter the number of measurement(s)");
	scanf("%d", &N_measure);
	printf("The number of measurement(s) is %d .\n", N_measure);
	puts("Enter the interval for measurement.");
	scanf("%d", &interval_measure);
	printf("The interval for measurement is %d .\n", interval_measure);
	puts("Enter the number of iterations for thermalization (must be divisibe by interval_measure).");
	scanf("%d", &N_thermal);
	if ( N_thermal%interval_measure!=0 )
	{
		puts("N_thermal is indivisible by interval_measure. Exit!");
		exit(1);
	}
	printf("The number of iterations for thermalization is %d .\n", N_thermal);
	puts("Enter the interval for display (must be divisible by interval_measure).");
	scanf("%d", &interval_display);
	if ( interval_display%interval_measure!=0 )
	{
		puts("interval_display is indivisible by interval_measure. Exit!");
		exit(1);
	}
	printf("The interval for display is %d .\n", interval_display);
	puts("Enter the interval for saving (must be divisible by interval_measure).");
	scanf("%d", &interval_save);
	if ( interval_save%interval_measure!=0 )
	{
		puts("N_thermal is indivisible by interval_measure. Exit!");
		exit(1);
	}
	printf("The interval for saving is %d .\n", interval_save);

	puts("Select the mode for simulation GPU/CPU/Both (g/c/b) .");
	scanf("%c", &mode);  // absorb the \n
	scanf("%c", &mode);
	if ( mode=='g' )
		puts("Do GPU simulation.");
	else if ( mode=='c' )
		puts("Do CPU simulation.");
	else if ( mode=='b' )
		puts("Do both GPU and CPU simulation.");
	else
	{
		puts("Must enter g, c or b! Exit!");
		exit(1);
	}
	printf("\n");

	if (mode!='c')
	{
		puts("Set the GPU ID.");
		scanf("%d", &gid);
		printf("GPU ID = %d is selected.\n");
		hipSetDevice(gid);
		hipEventCreate(&start);
		hipEventCreate(&stop);
	
		puts("Set the value m, n such that the theads per block is ( tx,ty ) = (2^m,2^n) (Nx must be divisible by tx and Ny must be divisible by ty).");
		scanf("%d %d", &m, &n);
		if ( m<1 )
		{
			puts("m must be positive! Exit!");
			exit(1);
		}
		else if ( n<1 )
		{
			puts("n must be positive! Exit!");
			exit(1);
		}
		tx = (int)pow(2.0, m);
		ty = (int)pow(2.0, n);
		if ( Nx%tx!=0 )
		{
			puts("Nx must be divisible by threads per block tx! Exit!");
			exit(1);
		}
		if ( Ny%ty!=0 )
		{
			puts("Ny must be divisible by threads per block ty! Exit!");
			exit(1);
		}
		printf("Threads per block is ( %d,%d ) .\n", tx, ty);
		puts("Use only one grids to accomdate the lattice so blocks per grid is auto-matically set.");
		bx = Nx/tx;
		if (bx%2!=0)
		{
			puts("bx must be even to apply double checkboard scheme for periodic boundary condition!");
			exit(1);
		}
		by = Ny/ty;
		if (by%2!=0)
		{
			puts("by must be even to apply double checkboard scheme for periodic boundary condition!");
			exit(1);
		}
		printf("Blocks per grid is ( %d,%d ) .\n", bx, by);
		printf("\n");

		int *d_spin;
		dim3 TpB (tx, ty);
		dim3 BpG (bx, by);
		N_block = bx*by;
		N_thread = tx*ty;
		hiprandState *devstate;

		hipMalloc((void**)&d_spin, size_site);

		total_time = 0.0;
		hipEventRecord(start, 0);

		// initialize the spin
		if (ini=='c')
			INITIALIZE_COLD<<<BpG, TpB>>> (d_spin);
		else
		{
			hipMalloc((void**)&devstate, N_site*sizeof(hiprandState));
			SET_UP_KERNEL<<<N_block, N_thread>>> ( seed, devstate);
			INITIALIZE_HOT<<<BpG, TpB>>> (d_spin, devstate);
	
			FILE *check = fopen("initial_spin_check.txt", "w");
			hipMemcpy(h_spin, d_spin, size_site, hipMemcpyDeviceToHost);

			for (int j=0; j<Ny; j++)
			{
				for (int i=0; i<Nx; i++)	
					fprintf(check, "%d\t", h_spin[i+j*Nx]);
				fprintf(check, "\n");
			}
			fprintf(check, "\n");
		}

		if (mode=='b')
			hipMemcpy(h_spin, d_spin, size_site, hipMemcpyDeviceToHost);
		//
	
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		total_time += gpu_time;
		printf("GPU time for initialize the spin lattice on device is %.4f ms.\n", gpu_time);		

//		int block_o, block_e;
		int thread_o, thread_e;
//		block_o = N_block/2;
//		block_e = (N_block+1)/2;
//		block_e = block_o;
		thread_o = N_thread/2;
//		thread_e = (N_thread+1)/2;
		thread_e = thread_o;
		int *odd_T_in_B, *even_T_in_B;
		hipMalloc((void**)&odd_T_in_B, 5*thread_o*sizeof(int *));
		hipMalloc((void**)&even_T_in_B, 5*thread_e*sizeof(int *));
		
		puts("Initialize the mapping from shared memory indices to thread indices.");
		hipEventRecord(start, 0);
		DOUBLE_CHECKERBOARD_SETUP <<<1, TpB>>> (odd_T_in_B, even_T_in_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		total_time += gpu_time;
		printf("GPU time for initializing the mapping from shared memory indices to thread indices is %.4f ms.\n", gpu_time);		
		
		double **exp_update = (double **)malloc(2*sizeof(double *));
		exp_update[0] = (double *)malloc(5*sizeof(double));
		exp_update[1] = (double *)malloc(2*sizeof(double));
		puts("Calculate the Boltzmann factor and store it in the constant memroy.");
		hipEventRecord(start, 0);
		UPDATE_MATRIX (exp_update[0], exp_update[1]);
//		for (int i=0; i<5; i++)
//			printf("%.4f\n", exp_update[0][i]);
//		for (int i=0; i<2; i++)
//			printf("%.4f\n", exp_update[1][i]);
		hipMemcpyToSymbol(HIP_SYMBOL(exp_update0_dev), exp_update[0], 5*sizeof(double));
		hipMemcpyToSymbol(HIP_SYMBOL(exp_update1_dev), exp_update[1], 2*sizeof(double));
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		total_time += gpu_time;
		printf("GPU time for calculating the Boltzmann factor and storing it in the constant memory is %.4f ms.\n", gpu_time);		
		
		//constant memory test
//		hipEventRecord(start, 0);
//		CONSTANT_MEMORY_TEST<<<1,1>>> ();
//		hipEventRecord(stop, 0);
//		hipEventSynchronize(stop);
//		hipEventElapsedTime(&gpu_time, start, stop);
//		printf("GPU time for calculating the Boltzmann factor and storing it in the constant memory test is %.4f ms.\n", gpu_time);		
		//
		//hiprand device API seed test
//		hipMalloc((void**)&devstate, sizeof(hiprandState));
//		SET_UP_KERNEL<<<N_block, N_thread>>> (seed, devstate);
//		for (int i=0; i<1; i++)
//			RANDOM_NUMBER_TEST<<<dim3(bx,by), dim3(tx,ty)>>> (i, devstate);
		//

		// simulation
		int size_shared = (tx+2)*(ty+2)*sizeof(int);
		int size_shared_measure = 2*N_thread*sizeof(int); // to calculate both M and E;
		int size_block = N_block*sizeof(int);
		int to_save = interval_save/interval_measure;
		int to_display = interval_display/interval_measure;
		double M_mean, E_mean, M_sigma, E_sigma;
		int *h_M, *h_E;
		int *d_M, *d_E;
		double *M_save, *E_save;
		FILE *output = fopen("spin_config_gpu.txt", "w");
		FILE *output2 = fopen("M_and_E_gpu.txt", "w");
		h_M = (int *)malloc(size_block);
		h_E = (int *)malloc(size_block);
		M_save = (double *)malloc(sizeof(double)*N_measure);
		E_save = (double *)malloc(sizeof(double)*N_measure);
		hipMalloc((void **)&d_M, size_block);
		hipMalloc((void **)&d_E, size_block);

		hipEventRecord(start, 0);
		hipMalloc((void**)&devstate, N_site/2/2*sizeof(hiprandState));
		SET_UP_KERNEL<<<N_block/2, N_thread/2>>> (seed, devstate);
			// thermalizatoin
		for (int N=0; N<N_thermal/interval_measure; N++)
		{
			ISING_ODD_BLOCK<<<dim3(bx/2,by), dim3(tx/2,ty), size_shared>>> (interval_measure, B, d_spin, devstate, odd_T_in_B, even_T_in_B);
			ISING_EVEN_BLOCK<<<dim3(bx/2,by), dim3(tx/2,ty), size_shared>>> (interval_measure, B, d_spin, devstate, odd_T_in_B, even_T_in_B);
		}
			//
			// measurement and updating
		M_mean = 0;
		E_mean = 0;
		M_sigma = 0;
		E_sigma = 0;
		puts("index of Meas.\t<M>\t\t<E>\n===============================================================");
		for (int N=0; N<N_measure; N++)
		{
				// measurement
			M_save[N] = 0;
			E_save[N] = 0;
			ISING_MEASUREMENT<<<BpG, TpB, size_shared_measure>>> (B, d_spin, d_M, d_E);
			hipMemcpy(h_M, d_M, size_block, hipMemcpyDeviceToHost);
			hipMemcpy(h_E, d_E, size_block, hipMemcpyDeviceToHost);
			for (int i=0; i<N_block; i++)
			{
				M_save[N] += h_M[i];
				E_save[N] += h_E[i];
			}			
//			M_mean += M_save[N];
			M_mean += abs(M_save[N]);
			E_mean += E_save[N];
				//
				//save
			if ((N+1)%to_save==0)
			{
				hipMemcpy(h_spin, d_spin, size_site, hipMemcpyDeviceToHost);
				fprintf(output, "#%d Spin Configuration\n===============================================================\n", (N+1)/to_save);
				for (int j=0; j<Ny; j++)
				{
					for (int i=0; i<Nx; i++)	
						fprintf(output, "%d\t", h_spin[i+j*Nx]);
					fprintf(output, "\n");
				}
				fprintf(output, "\n");
			}
				//
			if ((N+1)%to_display==0)
				printf("%d\t\t%.6f\t\t%.6f\n", N+1, M_mean/(N+1)/N_site, E_mean/(N+1)/N_site ); 
				//updating
			ISING_ODD_BLOCK<<<dim3(bx/2,by), dim3(tx/2,ty), size_shared>>> (interval_measure, B, d_spin, devstate, odd_T_in_B, even_T_in_B);
			ISING_EVEN_BLOCK<<<dim3(bx/2,by), dim3(tx/2,ty), size_shared>>> (interval_measure, B, d_spin, devstate, odd_T_in_B, even_T_in_B);
				//
		}

		puts("===============================================================");
		fprintf(output2, "<M>\t<E>\n===============================================================\n");
		M_mean /= (double)(N_site)*(double)(N_measure);
		E_mean /= (double)(N_site)*(double)(N_measure);
		for (int i=0; i<N_measure; i++)
		{
//			M_sigma += pow(M_save[i]/N_site-M_mean, 2.0);
			M_sigma += pow(abs(M_save[i])/N_site-M_mean, 2.0);
			E_sigma += pow(E_save[i]/N_site-E_mean, 2.0);
			fprintf(output2, "%.6f\t%.6f\n", M_save[i]/N_site, E_save[i]/N_site);
		}
		M_sigma = sqrt(M_sigma/(N_measure-1));
		E_sigma = sqrt(E_sigma/(N_measure-1));
		puts("GPU Simulation result:");
		printf("\tMagnetization: Mean = %.6f\t Stand Deviation = %.6f\n", M_mean, M_sigma);
		printf("\tEnergy: Mean = %.6f\t Stand Deviation = %.6f\n", E_mean, E_sigma);
			//
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&gpu_time, start, stop);
		total_time += gpu_time;
		printf("GPU time for simulation is %.4f ms.\n", gpu_time);		
		printf("Total GPU is %.4f ms.\n", total_time);		
		//
		free(exp_update);
		free(h_M);
		free(h_E);
		free(M_save);
		free(E_save);
		fclose(output);
		fclose(output2);
		hipFree(odd_T_in_B);
		hipFree(even_T_in_B);
		hipFree(d_spin);
		hipFree(d_M);
		hipFree(d_E);
		hipFree(devstate);

		if (mode=='g')
		{
			hipEventDestroy(start);
			hipEventDestroy(stop);
			hipDeviceReset();
		}
	}

	if (mode!='g')
	{
		rng = gsl_rng_alloc(gsl_rng_mt19937);
		gsl_rng_set(rng, seed);
		if (mode=='c')
		{
			hipEventCreate(&start);
			hipEventCreate(&stop);
		}
		// initialize the spin (for mode 'b', the spin is already initialized by GPU)
		if (ini=='c')
			for (int i=0; i<N_site; i++)
				h_spin[i] = 1;
		else
			for (int i=0; i<N_site; i++)
			{
				if (gsl_rng_uniform(rng)<=0.5)
					h_spin[i] = 1;
				else
					h_spin[i] = -1;
			}
		//
		hipEventRecord(start, 0);
		ISING_CPU();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&cpu_time, start, stop);
		printf("CPU time for simulation is %.4f ms.\n", cpu_time);		
		if (mode=='b')
			printf("Speed-up by GPU is: %.2f\n", cpu_time/total_time);
		hipDeviceReset();
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	if ( B==0.0 )
	{
		EXACT_EM(&E_exact, &M_exact);
		printf("Exact solution: Tc = %.6f , M_exact = %.6f, E_exact = %.6f .\n", Tc, M_exact, E_exact);
	}
	else
		puts("Exact solution when B is non-zero is waiting to be found!");

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipDeviceReset();
}

__device__ int UPDATE_SPIN_GPU(int old_spin, int new_spin, int spin_around, double x)
{
	if (old_spin==1)
	{
		if (spin_around==4)
		{
			if (x<exp_update0_dev[0])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==2)
		{
			if (x<exp_update0_dev[1])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==0)
		{
			if (x<exp_update0_dev[2])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==-2)
		{
			if (x<exp_update0_dev[3])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==-4)
		{
			if (x<exp_update0_dev[4])
				return new_spin;
			else
				return old_spin;
		}
		else
		{
			printf("Error happens for old_spin parallel to B! Exit!\n");
			printf("Spin aournd = %d\n", spin_around);
			return 0;
		}
	}
	else if (old_spin==-1)
	{
		if (spin_around==-4)
		{
			if (x<exp_update1_dev[0])
				return new_spin;
			else
				return old_spin;
		}
		else if (spin_around==-2)
		{
			if (x<exp_update1_dev[1])
				return new_spin;
			else
				return old_spin;
		}
		else
		{
			printf("Error happens for old spin anti-parallel to B! Exit!\n");
			printf("Spin aournd = %d\n", spin_around);
			return 0;
		}
	}
	else
	{
		printf("Error for old spin! Exit!\n");
		printf("Old spin = %d\n", old_spin);
		return 0;
	}
}

void EXACT_EM (double *E_exact, double *M_exact)
{
	double k = 1.0/sinh(2.0/T)/sinh(2.0/T);
	Tc =  2.0/log(1.0+sqrt(2.0));

	*E_exact = -1.0/tanh(2.0/T)*( 1.0 + 2.0/M_PI*(2.0*tanh(2.0/T)*tanh(2.0/T)-1.0)*gsl_sf_ellint_Kcomp(2.0*sqrt(k)/(1.0+k), GSL_PREC_DOUBLE) );

	if ( T<=Tc )
		*M_exact = pow( (1.0-pow(sinh(2.0/T), -4.0)), 0.125);
	else
		*M_exact = 0.0;
}

void UPDATE_MATRIX (double *A1, double *A2)
{
	A1[0] = exp(-2.0*(4.0+B)/T);
	A1[1] = exp(-2.0*(2.0+B)/T);
	A1[2] = exp(-2.0*B/T);
	A1[3] = exp(-2.0*(-2.0+B)/T);
	A1[4] = exp(-2.0*(-4.0+B)/T);

	A2[0] = exp(-2.0*(4.0-B)/T);
	A2[1] = exp(-2.0*(2.0-B)/T);
}

//void UPDATE_SPIN(int center, int old_spin, int new_spin, int spin_around, double delta_E, int *h_spin, double **exp_update, gsl_rng *rng)
void UPDATE_SPIN(int center, int old_spin, int new_spin, int spin_around, int delta_E, double **exp_update)
{
	if (delta_E<=0)
		h_spin[center] = new_spin;
	else if (delta_E>0)
	{
		if (old_spin==1)
		{
			if (spin_around==4)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][0])
					h_spin[center] = new_spin;
			}
			else if (spin_around==2)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][1])
					h_spin[center] = new_spin;
			}
			else if (spin_around==0)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][2])
					h_spin[center] = new_spin;
			}
			else if (spin_around==-2)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][3])
					h_spin[center] = new_spin;
			}
			else if (spin_around==-4)
			{
				if (gsl_rng_uniform(rng)<exp_update[0][4])
					h_spin[center] = new_spin;
			}
			else
			{
				puts("Error happens for old_spin parallel to B! Exit!");
				printf("Spin aournd = %d\n", spin_around);
				exit(1);
			}
		}
		else if (old_spin==-1)
		{
			if (spin_around==-4)
			{
				if (gsl_rng_uniform(rng)<exp_update[1][0])
					h_spin[center] = new_spin;
			}
			else if (spin_around==-2)
			{
				if (gsl_rng_uniform(rng)<exp_update[1][1])
					h_spin[center] = new_spin;
			}
			else
			{
				puts("Error happens for old spin anti-parallel to B! Exit!");
				printf("Spin aournd = %d\n", spin_around);
				exit(1);
			}
		}
		else
		{
			puts("Error for old spin! Exit!");
			printf("Old spin = %d\n", old_spin);
			exit(1);
		}
	} // end of delta_E>0 case
}

void ISING_CPU (void)
{
	int x, y, parity;
	int center, fw_x, bw_x, fw_y, bw_y, spin_around;
	int old_spin, new_spin, delta_E;
	int i_o = (int)(N_site/2);
	int i_e = (int)((N_site+1)/2);
	int *h_spin_odd = (int*)malloc(i_o*sizeof(int));
	int *h_spin_even = (int*)malloc(i_e*sizeof(int));
	double **exp_update = (double**)malloc(2*sizeof(double*));
	exp_update[0] = (double*)malloc(5*sizeof(double));
	exp_update[1] = (double*)malloc(2*sizeof(double));
//	gsl_rng rng;
//	rng = gsl_rng_alloc(gsl_rng_mt19937);
//	gsl_rng_set(rng, seed);
	UPDATE_MATRIX(exp_update[0], exp_update[1]);
	
	//odd site
	for (int i=0; i<i_o; i++)
	{
		x = (2*i)%Nx;
		y = (2*i)/Nx;
		parity = (x+y+1)%2;
		x = x+parity;
		h_spin_odd[i] = x+Nx*y;
	}
	//
	//even site
	for (int i=0; i<i_e; i++)
	{
		x = (2*i)%Nx;
		y = (2*i)/Nx;
		parity = (x+y)%2;
		x = x+parity;
		h_spin_even[i] = x+Nx*y;
	}
	//
	for (int N=0; N<N_thermal; N++)
	{
		for (int i=0; i<i_o; i++)
		{
			center = h_spin_odd[i];
			x = center%Nx;
			y = center/Nx;
			fw_x = (x+1)%Nx+y*Nx;
			bw_x = (x-1+Nx)%Nx+y*Nx;
			fw_y = x+((y+1)%Ny)*Nx;
			bw_y = x+((y-1+Ny)%Ny)*Nx;
			spin_around = h_spin[fw_x]+h_spin[bw_x]+h_spin[fw_y]+h_spin[bw_y];
			old_spin = h_spin[center];
			new_spin = -old_spin;
			delta_E = (old_spin-new_spin)*(spin_around+B); // delta_E = E_new-E_ole = -(spin_new[center]-spin_old[center])*(summation_of_neighboring_spin+B)
//			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, h_spin, exp_update, rng);
			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, exp_update);
		} // end of odd site
		for (int i=0; i<i_e; i++)
		{
			center = h_spin_even[i];
			x = center%Nx;
			y = center/Nx;
			fw_x = (x+1)%Nx+y*Nx;
			bw_x = (x-1+Nx)%Nx+y*Nx;
			fw_y = x+((y+1)%Ny)*Nx;
			bw_y = x+((y-1+Ny)%Ny)*Nx;
			spin_around = h_spin[fw_x]+h_spin[bw_x]+h_spin[fw_y]+h_spin[bw_y];
			old_spin = h_spin[center];
			new_spin = -old_spin;
			delta_E = (old_spin-new_spin)*(spin_around+B); // delta_E = E_new-E_ole = -(spin_new[center]-spin_old[center])*(summation_of_neighboring_spin+B)
//			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, h_spin, exp_update, rng);
			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, exp_update);
		} // end even_site
	} // end of thermalization

	int N = 0;
	int count = 1;
	double M_mean = 0, E_mean = 0;
	double M_sigma = 0, E_sigma = 0;
	double *M_save = (double*)malloc(N_measure*sizeof(double));
	double *E_save = (double*)malloc(N_measure*sizeof(double));
	FILE *output = fopen("spin_config_cpu.txt", "w");
	FILE *output2 = fopen("M_and_E_cpu.txt", "w");
	puts("index of Meas.\t<M>\t\t<E>\n===============================================================");
	while (N<N_measure)
	{
		if (count%interval_measure==0)
		{
			int M = 0, E = 0;
			for (int i=0; i<N_site; i++)
			{
				x = i%Nx;
				y = i/Nx;
				fw_x = (x+1)%Nx+y*Nx;
				fw_y = x+((y+1)%Ny)*Nx;
				spin_around = h_spin[fw_x]+h_spin[fw_y];
				M += h_spin[i];
				E += -h_spin[i]*(spin_around+B);
			}
//			M_mean += M;
			M_mean += abs(M);
			E_mean += E;
			M_save[N] = M;
			E_save[N] = E;
			N++;
		}

		if (count%interval_save==0)
		{
			fprintf(output, "#%d Spin Configuration\n===============================================================\n", (int)(count/interval_save));
			for (int j=0; j<Ny; j++)
			{
				for (int i=0; i<Nx; i++)	
					fprintf(output, "%d\t", h_spin[i+j*Nx]);
				fprintf(output, "\n");
			}
			fprintf(output, "\n");
		}
		
		if (count%interval_display==0)
			printf("%d\t\t%.6f\t\t%.6f\n", N, M_mean/N/N_site, E_mean/N/N_site ); 

		for (int i=0; i<i_o; i++)
		{
			center = h_spin_odd[i];
			x = center%Nx;
			y = center/Nx;
			fw_x = (x+1)%Nx+y*Nx;
			bw_x = (x-1+Nx)%Nx+y*Nx;
			fw_y = x+((y+1)%Ny)*Nx;
			bw_y = x+((y-1+Ny)%Ny)*Nx;
			spin_around = h_spin[fw_x]+h_spin[bw_x]+h_spin[fw_y]+h_spin[bw_y];
			old_spin = h_spin[center];
			new_spin = -old_spin;
			delta_E = (old_spin-new_spin)*(spin_around+B); // delta_E = E_new-E_ole = -(spin_new[center]-spin_old[center])*(summation_of_neighboring_spin+B)
//			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, h_spin, exp_update, rng);
			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, exp_update);
		} // end of odd site
		for (int i=0; i<i_e; i++)
		{
			center = h_spin_even[i];
			x = center%Nx;
			y = center/Nx;
			fw_x = (x+1)%Nx+y*Nx;
			bw_x = (x-1+Nx)%Nx+y*Nx;
			fw_y = x+((y+1)%Ny)*Nx;
			bw_y = x+((y-1+Ny)%Ny)*Nx;
			spin_around = h_spin[fw_x]+h_spin[bw_x]+h_spin[fw_y]+h_spin[bw_y];
			old_spin = h_spin[center];
			new_spin = -old_spin;
			delta_E = (old_spin-new_spin)*(spin_around+B); // delta_E = E_new-E_ole = -(spin_new[center]-spin_old[center])*(summation_of_neighboring_spin+B)
//			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, h_spin, exp_update, rng);
			UPDATE_SPIN(center, old_spin, new_spin, spin_around, delta_E, exp_update);
		} // end even_site
		count++;
	} // end of measurements
	puts("===============================================================");
	fprintf(output2, "<M>\t<E>\n===============================================================\n");
	M_mean /= (N_site*N_measure);
	E_mean /= (N_site*N_measure);
	for (int i=0; i<N_measure; i++)
	{
//		M_sigma += pow(M_save[i]/N_site-M_mean, 2.0);
		M_sigma += pow(abs(M_save[i])/N_site-M_mean, 2.0);
		E_sigma += pow(E_save[i]/N_site-E_mean, 2.0);
		fprintf(output2, "%.6f\t%.6f\n", M_save[i]/N_site, E_save[i]/N_site);
	}
	M_sigma = sqrt(M_sigma/(N_measure-1));
	E_sigma = sqrt(E_sigma/(N_measure-1));
	puts("CPU Simulation result:");
	printf("\tMagnetization: Mean = %.6f\t Stand Deviation = %.6f\n", M_mean, M_sigma);
	printf("\tEnergy: Mean = %.6f\t Stand Deviation = %.6f\n", E_mean, E_sigma);

	free(h_spin_odd);
	free(h_spin_even);
	free(exp_update);
	free(M_save);
	free(E_save);
	fclose(output);
	fclose(output2);
} // end of ISING_CPU
