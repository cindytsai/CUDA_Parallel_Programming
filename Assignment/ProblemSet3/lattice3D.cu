// Solve the Laplace equation on a 2D lattice with boundary conditions.
//
// compile with the following command:
//
// (for GTX970)
// nvcc -arch=compute_52 -code=sm_52,sm_52 -O3 -m64 -o laplace laplace.cu
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O3 -m64 -o laplace laplace.cu


// Includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// field variables
float* h_new;   // host field vectors
float* h_old;   
float* h_C;     // result of diff*diff of each block
float* g_new;   
float* d_new;   // device field vectors
float* d_old;  
float* d_C;

int     MAX=1000000;      // maximum iterations
double  eps=1.0e-10;      // stopping criterion


__global__ void laplacian(float* phi_old, float* phi_new, float* C, bool flag)
{
    extern __shared__ float cache[];     
    float  t, l, r, b, u, d;     // top, left, right, bottom, up, down
    float  diff; 
    int    site, ym1, xm1, zm1, xp1, yp1, zp1;

    int Nx = blockDim.x*gridDim.x; // number of site in x direction
    int Ny = blockDim.y*gridDim.y; // number of site in y direction
    int Nz = blockDim.z*gridDim.z; // number of site in z direction
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    int z = blockDim.z*blockIdx.z + threadIdx.z;
    int cacheIndex = threadIdx.x + threadIdx.y*blockDim.x + threadIdx.z*blockDim.x*blockDim.y;  

    site = x + y*Nx + z*Nx*Ny;

    if((x == 0) || (x == Nx-1) || (y == 0) || (y == Ny-1) || (z == 0) || (z == Nz - 1)) {  
      // Do nothing on the boundary
    }
    else {
      xm1 = site - 1;    // x-1
      xp1 = site + 1;    // x+1
      ym1 = site - Nx;   // y-1
      yp1 = site + Nx;   // y+1
      zm1 = site - Nx*Ny; // z-1
      zp1 = site + Nx*Ny; // z+1
      if(flag) {
        b = phi_old[ym1]; 
        l = phi_old[xm1];
        d = phi_old[zm1]; 
        r = phi_old[xp1]; 
        t = phi_old[yp1];
        u = phi_old[zp1];
        phi_new[site] = (1.0/6.0)*(b + l + r + t + u + d);
      }
      else {
        b = phi_new[ym1]; 
        l = phi_new[xm1];
        d = phi_new[zm1]; 
        r = phi_new[xp1]; 
        t = phi_new[yp1];
        u = phi_new[zp1];
        phi_old[site] = (1.0/6.0)*(b + l + r + t + u + d);
      }
      diff = phi_new[site]-phi_old[site];
    }
    cache[cacheIndex]=diff*diff;
    __syncthreads();

    // perform parallel reduction

    int ib = (blockDim.x * blockDim.y * blockDim.z)/ 2;  
    while (ib != 0) {  
      if(cacheIndex < ib)  
        cache[cacheIndex] += cache[cacheIndex + ib];
      __syncthreads();
      ib /=2;  
    } 
    int blockIndex = blockIdx.x + gridDim.x*blockIdx.y + gridDim.x*gridDim.y*blockIdx.z;
    if(cacheIndex == 0)  C[blockIndex] = cache[0];
}

int main(void)
{

    int gid;              // GPU_ID
    int iter;
    volatile bool flag;   // to toggle between *_new and *_old  
    float cputime;
    float gputime;
    float gputime_tot;
    double error;
    
    printf("Enter the GPU ID (0/1): ");
    scanf("%d",&gid);
    printf("%d\n",gid);

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Select GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("Solve Laplace equation on a 3D lattice with boundary conditions\n");

    int Nx, Ny, Nz;                // lattice size
    printf("Enter the size (Nx, Ny, Nz) of the 3D lattice: ");
    scanf("%d %d %d", &Nx, &Ny, &Nz);        
    printf("%d %d %d\n", Nx, Ny, Nz);        

    // Set the number of threads (tx,ty,tz) per block
   
    int tx, ty, tz;
    printf("Enter the number of threads (tx, ty, tz) per block: ");
    scanf("%d %d %d", &tx, &ty, &tz);
    printf("%d %d %d\n", tx, ty, tz);
    if( tx > 1024 ) {
      printf("Max dimension size of a thread block (x,y,z): (1024, 1024, 64)\n");
      exit(0);
    }
    if( ty > 1024 ) {
      printf("Max dimension size of a thread block (x,y,z): (1024, 1024, 64)\n");
      exit(0);
    }
    if( tz > 64 ) {
      printf("Max dimension size of a thread block (x,y,z): (1024, 1024, 64)\n");
      exit(0);
    }
    if( tx * ty * tz > 1024 ) {
      printf("The number of threads per block must be less than 1024 ! \n");
      exit(0);
    }
    dim3 threads(tx, ty, tz); 
    
    // The total number of threads in the grid is equal to the total number of lattice sites
    
    int bx = Nx / tx;
    if(bx * tx != Nx) {
      printf("The block size in x is incorrect\n"); 
      exit(0);
    }
    int by = Ny / ty;
    if(by * ty != Ny) {
      printf("The block size in y is incorrect\n"); 
      exit(0);
    }
    int bz = Nz / tz;
    if(bz * tz != Nz) {
      printf("The block size in z is incorrect\n"); 
      exit(0);
    }
    if((bx > 2147483647) || (by > 65535) || (bz > 65535)) {
      printf("The grid size exceeds the limit ! \n");
      exit(0);
    }
    dim3 blocks(bx, by, bz);
    printf("The dimension of the grid is (%d, %d, %d)\n", bx, by, bz); 

    int CPU;    
    printf("To compute the solution vector with CPU/GPU/both (0/1/2) ? ");
    scanf("%d",&CPU);
    printf("%d\n",CPU);
    fflush(stdout);

    // Allocate field vector h_phi in host memory

    int N = Nx * Ny * Nz;
    int size = N * sizeof(float);
    int sb = bx * by * bz * sizeof(float);
    h_old = (float*)malloc(size);
    h_new = (float*)malloc(size);
    g_new = (float*)malloc(size);
    h_C = (float*)malloc(sb);
   
    // Initialize the array to 0
    memset(h_old, 0, size);
    memset(h_new, 0, size);

    // Initialize the field vector with boundary conditions
    for(int x = 0; x < Nx; x = x+1) {
      for(int y = 0; y < Ny; y = y+1){
        h_new[x + Nx*y + Nx*Ny*(Nz-1)]=1.0; 
        h_old[x + Nx*y + Nx*Ny*(Nz-1)]=1.0;
      }
    }  

    FILE *out1;                 // save initial configuration in phi_initial.dat
    out1 = fopen("phi_initial_3D.dat","w");
    for(int k = Nz-1; k > -1; k = k-1){
      fprintf(out1, "z = %d\n", k);
      for(int j = Ny-1; j > -1; j = j-1){
        for(int i = 0; i < Nx; i = i+1){
          fprintf(out1,"%.2e ",h_new[i + j*Nx + k*Nx*Ny]);
        }
        fprintf(out1,"\n");
      }
    }
    fclose(out1);

    printf("\n");

    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if(CPU>0) {

      // start the timer
      hipEventRecord(start,0);

      // Allocate vectors in device memory

      hipMalloc((void**)&d_new, size);
      hipMalloc((void**)&d_old, size);
      hipMalloc((void**)&d_C, sb);
  
      // Copy vectors from host memory to device memory

      hipMemcpy(d_new, h_new, size, hipMemcpyHostToDevice);
      hipMemcpy(d_old, h_old, size, hipMemcpyHostToDevice);
    
      // stop the timer
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      float Intime;
      hipEventElapsedTime( &Intime, start, stop);
      printf("Input time for GPU: %f (ms) \n",Intime);

      // start the timer
      hipEventRecord(start,0);

      error = 10*eps;  // any value bigger than eps is OK
      iter = 0;        // counter for iterations
      flag = true; 
 
      int sm = tx * ty * tz * sizeof(float);   // size of the shared memory in each block

      while ( (error > eps) && (iter < MAX) ) {
        laplacian<<<blocks, threads, sm>>>(d_old, d_new, d_C, flag);
        hipMemcpy(h_C, d_C, sb, hipMemcpyDeviceToHost);
        error = 0.0;
        for(int i = 0; i < bx * by * bz; i = i+1) {
          error = error + h_C[i];
        }
        error = sqrt(error);

        iter++;
        flag = !flag;
      }

      printf("error (GPU) = %.15e\n",error);
      printf("total iterations (GPU) = %d\n",iter);
    
      // stop the timer
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      hipEventElapsedTime( &gputime, start, stop);
      printf("Processing time for GPU: %f (ms) \n",gputime);
      
    
      // Copy result from device memory to host memory
  
      // start the timer
      hipEventRecord(start,0);

      // Because after the iteration, d_new and d_old are basically the same.
      hipMemcpy(g_new, d_new, size, hipMemcpyDeviceToHost);

      hipFree(d_new);
      hipFree(d_old);
      hipFree(d_C);

      // stop the timer
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      float Outime;
      hipEventElapsedTime( &Outime, start, stop);
      printf("Output time for GPU: %f (ms) \n",Outime);

      gputime_tot = Intime + gputime + Outime;
      printf("Total time for GPU: %f (ms) \n",gputime_tot);
      fflush(stdout);

      FILE *outg;                 // save GPU solution in phi_GPU.dat
      outg = fopen("phi_GPU_3D.dat","w");
      for(int k = Nz-1; k > -1; k = k-1){
        fprintf(outg, "z = %d\n", k);
        for(int j = Ny-1; j > -1; j = j-1){
          for(int i = 0; i < Nx; i = i+1){
            fprintf(outg, "%.2e ",g_new[i + j*Nx + k*Nx*Ny]);
          }
          fprintf(outg,"\n");
        }
      }
      fclose(outg);

      printf("\n");
    } 

    if(CPU==1) {      // not to compute the CPU solution 
      free(h_new);
      free(h_old);
      free(g_new);
      free(h_C);
      hipDeviceReset();
      exit(0);
    }
 
    if((CPU==0)||(CPU==2)) {      // to compute the CPU solution 

      // start the timer
      hipEventRecord(start,0);

      // to compute the reference solution

      error = 10*eps;      // any value bigger than eps 
      iter = 0;            // counter for iterations
      flag = true;     
      double diff; 

      float t, l, r, b, u, d;    // top, left, right, bottom, up, down
      int site, ym1, xm1, zm1, xp1, yp1, zp1;

      while ( (error > eps) && (iter < MAX) ) {
        if(flag) {
          error = 0.0;
          for(int z = 0; z < Nz; z = z+1){
            for(int y = 0; y < Ny; y = y+1) {
              for(int x = 0; x < Nx; x = x+1) { 
                if(x==0 || x==Nx-1 || y==0 || y==Ny-1 || z==0 || z==Nz-1) {   
                  // Do nothing on the boundary
                }
                else {
                  site = x + y*Nx + z*Nx*Ny;
                  xm1 = site - 1;     // x-1
                  xp1 = site + 1;     // x+1
                  ym1 = site - Nx;    // y-1
                  yp1 = site + Nx;    // y+1
                  zm1 = site - Nx*Ny; // z-1
                  zp1 = site + Nx*Ny; // z+1
                  b = h_old[ym1]; 
                  l = h_old[xm1];
                  d = h_old[zm1]; 
                  r = h_old[xp1]; 
                  t = h_old[yp1];
                  u = h_old[zp1];
                  h_new[site] = (1.0/6.0)*(b + l + r + t + u + d);
                  diff = h_new[site]-h_old[site]; 
                  error = error + diff*diff;
                }
              } 
            } 
          }
        }
        else{
          error = 0.0;
          for(int z = 0; z < Nz; z = z+1){
            for(int y = 0; y < Ny; y = y+1) {
              for(int x = 0; x < Nx; x = x+1) { 
                if(x==0 || x==Nx-1 || y==0 || y==Ny-1 || z==0 || z==Nz-1) {   
                  // Do nothing on the boundary
                }
                else {
                  site = x + y*Nx + z*Nx*Ny;
                  xm1 = site - 1;     // x-1
                  xp1 = site + 1;     // x+1
                  ym1 = site - Nx;    // y-1
                  yp1 = site + Nx;    // y+1
                  zm1 = site - Nx*Ny; // z-1
                  zp1 = site + Nx*Ny; // z+1
                  b = h_new[ym1]; 
                  l = h_new[xm1];
                  d = h_new[zm1]; 
                  r = h_new[xp1]; 
                  t = h_new[yp1];
                  u = h_new[zp1];
                  h_old[site] = (1.0/6.0)*(b + l + r + t + u + d);
                  diff = h_new[site]-h_old[site]; 
                  error = error + diff*diff;
                }
              } 
            } 
          }
        }        

        flag = !flag;
        iter = iter + 1;
        error = sqrt(error);
      }                   // exit if error < eps
    
      printf("error (CPU) = %.15e\n",error);
      printf("total iterations (CPU) = %d\n",iter);

      // stop the timer
      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      hipEventElapsedTime( &cputime, start, stop);
      printf("Processing time for CPU: %f (ms) \n",cputime);

      printf("Speed up of GPU = %f\n", cputime/(gputime_tot));
      fflush(stdout);

      // destroy the timer
      hipEventDestroy(start);
      hipEventDestroy(stop);

      FILE *outc;                 // save CPU solution in phi_CPU.dat
      outc = fopen("phi_CPU_3D.dat","w");
      for(int k = Nz-1; k > -1; k = k-1){
        fprintf(outc, "z = %d\n", k);
        for(int j = Ny-1; j > -1; j = j-1){
          for(int i = 0; i < Nx; i = i+1){
            fprintf(outc, "%.2e ",h_new[i + j*Nx + k*Nx*Ny]);
          }
          fprintf(outc,"\n");
        }
      }
      fclose(outc);

      printf("\n");

      free(h_new);
      free(h_old);
      free(g_new);
      free(h_C);

    }

    hipDeviceReset();
    
}

