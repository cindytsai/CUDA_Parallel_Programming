#include "hip/hip_runtime.h"
// Solve the Laplace equation on a 2D lattice with boundary conditions.
// (using texture memory)
//
// compile with the following command:
//
// (for GTX970)
// nvcc -arch=compute_52 -code=sm_52,sm_52 -O3 -m64 -o laplace laplace.cu
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O3 -m64 -o laplace laplace.cu


// Includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// field variables
float* h_new;   // host field vectors
float* h_old;   
float* h_C;     // sum of diff*diff of each block
float* g_new;   // device solution back to the host 
float* d_new;   // device field vectors
float* d_old;  
float* d_C;     // sum of diff*diff of each block 

int     MAX=1000000;      // maximum iterations
double  eps=1.0e-10;      // stopping criterion


__align__(8) texture<float>  texOld;   // declare the texture
__align__(8) texture<float>  texNew;


__global__ void laplacian(float* phi_old, float* phi_new, float* C, bool flag)
{
    extern __shared__ float cache[];     
    float  t, l, c, r, b;     // top, left, center, right, bottom
    float  diff; 
    int    site, ym1, xm1, xp1, yp1;    

    int Nx = blockDim.x*gridDim.x;
    int Ny = blockDim.y*gridDim.y;
    int x = blockDim.x*blockIdx.x + threadIdx.x;
    int y = blockDim.y*blockIdx.y + threadIdx.y;
    int cacheIndex = threadIdx.x + threadIdx.y*blockDim.x;  

    site = x + y*Nx;

    if((x == 0) || (x == Nx-1) || (y == 0) || (y == Ny-1) ) {  
      // do nothing on the boundaries 
    }
    else {
      xm1 = site - 1;    // x-1
      xp1 = site + 1;    // x+1
      ym1 = site - Nx;   // y-1
      yp1 = site + Nx;   // y+1
      if(flag) {
        b = tex1Dfetch(texOld, ym1);      // read d_old via texOld
        l = tex1Dfetch(texOld, xm1);
        c = tex1Dfetch(texOld, site);
        r = tex1Dfetch(texOld, xp1);
        t = tex1Dfetch(texOld, yp1);
        phi_new[site] = 0.25*(b+l+r+t);
        diff = phi_new[site]-c;
      }
      else {
        b = tex1Dfetch(texNew, ym1);     // read d_new via texNew
        l = tex1Dfetch(texNew, xm1);
        c = tex1Dfetch(texNew, site);
        r = tex1Dfetch(texNew, xp1);
        t = tex1Dfetch(texNew, yp1);
        phi_old[site] = 0.25*(b+l+r+t);
        diff = phi_old[site]-c;
      }
    }

    // each thread saves its error estimate to the shared memory

    cache[cacheIndex]=diff*diff;  
    __syncthreads();

    // parallel reduction in each block 

    int ib = blockDim.x*blockDim.y/2;  
    while (ib != 0) {  
      if(cacheIndex < ib)  
        cache[cacheIndex] += cache[cacheIndex + ib];
      __syncthreads();
      ib /=2;  
    } 

    // save the partial sum of each block to C

    int blockIndex = blockIdx.x + gridDim.x*blockIdx.y;
    if(cacheIndex == 0)  C[blockIndex] = cache[0];    
}


int main(void)
{

    double  error;            // error estimate 

    int gid;

    printf("Enter the GPU ID (0/1): ");
    scanf("%d",&gid);
    printf("%d\n",gid);

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    hipSetDevice(gid);
    printf("Select GPU with device ID = %d\n", gid);

    printf("Solve Laplace equation on a 2D lattice with boundary conditions (using texture)\n");

    int Nx,Ny;                // lattice size
    printf("Enter the size (Nx, Ny) of the 2D lattice: ");
    scanf("%d %d",&Nx,&Ny);        
    printf("%d %d\n",Nx,Ny);        

    // Set the number of threads (tx,ty) per block
   
    int tx,ty;
    printf("Enter the number of threads (tx,ty) per block: ");
    scanf("%d %d",&tx, &ty);
    printf("%d %d\n",tx, ty);
    if( tx*ty > 1024 ) {
      printf("The number of threads per block must be less than 1024 ! \n");
      exit(0);
    }
    dim3 threads(tx,ty); 
    
    // The total number of threads in the grid is equal to the total number of lattice sites
    
    int bx = Nx/tx;
    if(bx*tx != Nx) {
      printf("The blocksize in x is incorrect\n"); 
      exit(0);
    }
    int by = Ny/ty;
    if(by*ty != Ny) {
      printf("The blocksize in y is incorrect\n"); 
      exit(0);
    }
    if((bx > 65535)||(by > 65535)) {
      printf("The grid size exceeds the limit ! \n");
      exit(0);
    }
    dim3 blocks(bx,by);
    printf("The dimension of the grid is (%d, %d)\n",bx,by); 

    int CPU;    
    printf("To compute the solution vector with CPU (1/0) ?");
    scanf("%d",&CPU);
    printf("%d\n",CPU);
    fflush(stdout);
   
    // Allocate field vector h_phi in host memory

    int N = Nx*Ny;
    int size = N*sizeof(float);
    int sb = bx*by*sizeof(float);
    h_old = (float*)malloc(size);
    h_new = (float*)malloc(size);
    g_new = (float*)malloc(size);
    h_C = (float*)malloc(sb);
   
    memset(h_old, 0, size);    
    memset(h_new, 0, size);

    // Initialize the field vector with boundary conditions

    for(int x=0; x<Nx; x++) {
      h_new[x+Nx*(Ny-1)]=1.0;  
      h_old[x+Nx*(Ny-1)]=1.0;
    }  

    FILE *out1;		        // save initial configuration in phi_initial.dat 
    out1 = fopen("phi_initial.dat","w");

    fprintf(out1, "Inital field configuration:\n");
    for(int j=Ny-1;j>-1;j--) {
      for(int i=0; i<Nx; i++) {
        fprintf(out1,"%.2e ",h_new[i+j*Nx]);
      }
      fprintf(out1,"\n");
    }
    fclose(out1);

//   printf("\n");
//    printf("Inital field configuration:\n");
//    for(int j=Ny-1;j>-1;j--) {
//      for(int i=0; i<Nx; i++) {
//        printf("%.2e ",h_new[i+j*Nx]);
//      }
//      printf("\n");
//    }
//    printf("\n");

    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start,0);

    // Allocate vectors in device memory

    hipMalloc((void**)&d_new, size);
    hipMalloc((void**)&d_old, size);
    hipMalloc((void**)&d_C, sb);

    hipBindTexture(NULL, texOld, d_old, size);   // bind the texture to already existed variable on 
    hipBindTexture(NULL, texNew, d_new, size);   // device memory

    // Copy vectors from host memory to device memory

    hipMemcpy(d_new, h_new, size, hipMemcpyHostToDevice);
    hipMemcpy(d_old, h_old, size, hipMemcpyHostToDevice);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Intime;
    hipEventElapsedTime( &Intime, start, stop);
    printf("Input time for GPU: %f (ms) \n",Intime);

    // start the timer
    hipEventRecord(start,0);

    error = 10*eps;      // any value bigger than eps is OK
    int iter = 0;        // counter for iterations
    double diff; 

    volatile bool flag = true;     

    int sm = tx*ty*sizeof(float);   // size of the shared memory in each block

    while ( (error > eps) && (iter < MAX) ) {

      laplacian<<<blocks,threads,sm>>>(d_old, d_new, d_C, flag);
      hipMemcpy(h_C, d_C, sb, hipMemcpyDeviceToHost);
      error = 0.0;
      for(int i=0; i<bx*by; i++) {
        error = error + h_C[i];
      }
      error = sqrt(error);

//      printf("error = %.15e\n",error);
//      printf("iteration = %d\n",iter);

      iter++;
      flag = !flag;
      
    }
     
    printf("error (GPU) = %.15e\n",error);
    printf("total iterations (GPU) = %d\n",iter);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float gputime;
    double flops;
    hipEventElapsedTime( &gputime, start, stop);
    printf("Processing time for GPU: %f (ms) \n",gputime);
    flops = 7.0*(Nx-2)*(Ny-2)*iter;
    printf("GPU Gflops: %f\n",flops/(1000000.0*gputime));
    
    // Copy result from device memory to host memory

    // start the timer
    hipEventRecord(start,0);

    hipMemcpy(g_new, d_new, size, hipMemcpyDeviceToHost);

    hipFree(d_new);
    hipFree(d_old);
    hipFree(d_C);

    hipUnbindTexture(texOld);
    hipUnbindTexture(texNew);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Outime;
    hipEventElapsedTime( &Outime, start, stop);
    printf("Output time for GPU: %f (ms) \n",Outime);

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n",gputime_tot);
    fflush(stdout);

    printf("\n");
//    printf("Final field configuration (GPU):\n");
//    for(int j=Ny-1;j>-1;j--) {
//      for(int i=0; i<Nx; i++) {
//        printf("%.2e ",g_new[i+j*Nx]);
//      }
//      printf("\n");
//    }
//    printf("\n");

    FILE *outg;                 // save GPU solution in phi_GPU_Tex.dat 
    outg = fopen("phi_GPU_Tex.dat","w");

    fprintf(outg, "GPU (using texture) field configuration:\n");
    for(int j=Ny-1;j>-1;j--) {
      for(int i=0; i<Nx; i++) {
        fprintf(outg,"%.2e ",g_new[i+j*Nx]);
      }
      fprintf(outg,"\n");
    }
    fclose(outg);

    // start the timer
    hipEventRecord(start,0);

    if(CPU==0) {
      free(h_new);
      free(h_old);
      free(g_new);
      free(h_C);
      hipDeviceReset();
    } 
 
    // to compute the reference solution

    error = 10*eps;      // any value bigger than eps 
    iter = 0;            // counter for iterations
    flag = true;     

    float t, l, r, b;    // top, left, right, bottom
    int site, ym1, xm1, xp1, yp1;

    while ( (error > eps) && (iter < MAX) ) {
      if(flag) {
        error = 0.0;
        for(int y=0; y<Ny; y++) {
        for(int x=0; x<Nx; x++) { 
          if(x==0 || x==Nx-1 || y==0 || y==Ny-1) {   
          }
          else {
            site = x+y*Nx;
            xm1 = site - 1;    // x-1
            xp1 = site + 1;    // x+1
            ym1 = site - Nx;   // y-1
            yp1 = site + Nx;   // y+1
            b = h_old[ym1]; 
            l = h_old[xm1]; 
            r = h_old[xp1]; 
            t = h_old[yp1]; 
            h_new[site] = 0.25*(b+l+r+t);
            diff = h_new[site]-h_old[site]; 
            error = error + diff*diff;
          }
        } 
        } 
      }
      else {
        error = 0.0;
        for(int y=0; y<Ny; y++) {
        for(int x=0; x<Nx; x++) { 
          if(x==0 || x==Nx-1 || y==0 || y==Ny-1) {
          }
          else {
            site = x+y*Nx;
            xm1 = site - 1;    // x-1
            xp1 = site + 1;    // x+1
            ym1 = site - Nx;   // y-1
            yp1 = site + Nx;   // y+1
            b = h_new[ym1]; 
            l = h_new[xm1]; 
            r = h_new[xp1]; 
            t = h_new[yp1]; 
            h_old[site] = 0.25*(b+l+r+t);
            diff = h_new[site]-h_old[site]; 
            error = error + diff*diff;
          } 
        }
        }
      }
      flag = !flag;
      iter++;
      error = sqrt(error);

//      printf("error = %.15e\n",error);
//      printf("iteration = %d\n",iter);

    }   // exit if error < eps
    
    printf("error (CPU) = %.15e\n",error);
    printf("total iterations (CPU) = %d\n",iter);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    flops = 7.0*(Nx-2)*(Ny-2)*iter;
    printf("CPU Gflops: %lf\n",flops/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/(gputime_tot));
    fflush(stdout);

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    FILE *outc;               // save CPU solution in phi_CPU.dat 
    outc = fopen("phi_CPU.dat","w");

    fprintf(outc,"CPU field configuration:\n");
    for(int j=Ny-1;j>-1;j--) {
      for(int i=0; i<Nx; i++) {
        fprintf(outc,"%.2e ",h_new[i+j*Nx]);
      }
      fprintf(outc,"\n");
    }
    fclose(outc);

//    printf("\n");
//    printf("Final field configuration (CPU):\n");
//    for(int j=Ny-1;j>-1;j--) {
//      for(int i=0; i<Nx; i++) {
//        printf("%.2e ",h_new[i+j*Nx]);
//      }
//      printf("\n");
//    }

    free(h_new);
    free(h_old);
    free(h_C);
    free(g_new);

    hipDeviceReset();
}

