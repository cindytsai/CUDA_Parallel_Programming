// To compute histogram with atomic operations */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


// Variables
float* data_h;          // host vectors
unsigned int* hist_h;   // GPU solution back to the CPU 
float* data_d;          // device vectors
unsigned int* hist_d;
unsigned int* hist_c;   // CPU solution


// Functions
void RandomUniform(float*, long);
void RandomNormal(float*, long);
void RandomExpDecay(float*, long);

__global__ void hist_shmem(float *data, const long N, unsigned int *hist, 
                           const int bins, const float Rmin, const float binsize) 
{

    // use shared memory and atomic addition

    extern __shared__  unsigned int temp[];     // assume the blocksize is equal to the total # bins
    temp[threadIdx.x] = 0;
    __syncthreads();

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    long stride = blockDim.x * gridDim.x;

//    if( (index > bins-1) || (index < 0)) {
//      printf("data[%d]=%f, index=%d\n",i,data[i],index);
//    }

    while (i < N) {
        int index = (int)((data[i] - Rmin) / binsize);
        atomicAdd(&temp[index], 1);
        i += stride;
    }

    __syncthreads();
    atomicAdd( &(hist[threadIdx.x]), temp[threadIdx.x] );

}


int main(void)
{

    int gid;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    scanf("%d",&gid);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Set GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("To find the histogram of a data set (with real numbers): \n");
    long N; 
    int bins,index;
    float Rmin, Rmax, binsize;

    printf("Enter the size of the data vector: ");
    scanf("%ld",&N);
    printf("%ld\n",N);
    long size = N * sizeof(float);

    printf("Enter the data range [Rmin, Rmax] for the histogram: ");
    scanf("%f %f",&Rmin, &Rmax);
    printf("%f %f\n",Rmin, Rmax);

    printf("Enter the number of bins of the histogram: ");
    scanf("%d",&bins);
    printf("%d\n",bins);
    if(bins > 1024) {
        printf("The number of bins is set to # of threads per block < 1024 ! \n");
        exit(0);
    }
    int bsize = bins*sizeof(int);
    binsize = (Rmax - Rmin)/(float)bins;
     
    data_h = (float*)malloc(size);
    hist_h = (unsigned int*)malloc(bsize);

    // Check memory allocations
    if(data_h == NULL || hist_h == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    for(int i=0; i<bins; i++){
        hist_h[i]=0;
    }

    // initialize the data_h vector
    // srand(time(NULL));  // initialize the seed with the current time
    srand(12345);  

//    RandomUniform(data_h, N);      // uniform deviate in (0,1)
    RandomExpDecay(data_h, N);       

    int threadsPerBlock;
    printf("Enter the number of threads per block: ");
    scanf("%d",&threadsPerBlock);
    printf("%d\n",threadsPerBlock);
    if( threadsPerBlock != bins ) {
        printf("The number of threads per block must be equal to the number of bins ! \n");
        exit(0);
    }
    fflush(stdout);

    int blocksPerGrid;
    printf("Enter the number of blocks per grid: ");
    scanf("%d",&blocksPerGrid);
    printf("%d\n",blocksPerGrid);
    if( blocksPerGrid > 2147483647 ) {
        printf("The number of blocks must be less than 2147483647 ! \n");
        exit(0);
    }
    printf("The number of blocks is %d\n", blocksPerGrid);
    fflush(stdout);

    int CPU;
    printf("To compute the histogram with CPU (1/0) ? ");
    scanf("%d",&CPU);
    printf("%d\n",CPU);
    fflush(stdout);


    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start,0);

    // Allocate vectors in device memory

    hipMalloc((void**)&hist_d, bsize);
    hipMalloc((void**)&data_d, size);

    // Copy vectors from host memory to device memory

    hipMemcpy(data_d, data_h, size, hipMemcpyHostToDevice);
    hipMemcpy(hist_d, hist_h, bsize, hipMemcpyHostToDevice);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Intime;
    hipEventElapsedTime( &Intime, start, stop);
    printf("Input time for GPU: %f (ms) \n",Intime);

   // start the timer
    hipEventRecord(start,0);
    
    int sm = threadsPerBlock * sizeof(int);

    hist_shmem <<< blocksPerGrid, threadsPerBlock, sm >>> (data_d, N, hist_d, bins, Rmin, binsize);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float gputime;
    hipEventElapsedTime( &gputime, start, stop);
    printf("Processing time for GPU: %f (ms) \n",gputime);
    printf("GPU Gflops: %f\n",2*N/(1000000.0*gputime));

    // Copy result from device memory to host memory
    // hist_h contains the result in host memory

    // start the timer
    hipEventRecord(start,0);

    hipMemcpy(hist_h, hist_d, bsize, hipMemcpyDeviceToHost);

    hipFree(data_d);
    hipFree(hist_d);

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Outime;
    hipEventElapsedTime( &Outime, start, stop);
    printf("Output time for GPU: %f (ms) \n", Outime);

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n", gputime_tot);

    // Save histogram in file
    FILE *out;
    out = fopen("hist_shmem.dat","w");

    fprintf(out, "Histogram (GPU):\n");
    for(int i=0; i<bins; i++) {
        float x = Rmin + (i + 0.5) * binsize;         // the center of each bin
        fprintf(out,"%f %d \n",x,hist_h[i]);
    }
    fclose(out);

    // Print the histogram on screen
    printf("Histogram (GPU):\n");
    for(int i = 0; i < bins; i = i+1) {
      float x = Rmin + (i + 0.5) * binsize;         // the center of each bin
      printf("%f %d\n", x, hist_h[i]);
    }

    if(CPU==0) {
      hipEventDestroy(start);
      hipEventDestroy(stop);
      hipDeviceReset();
      free(data_h);
      free(hist_h);
      return 0;
    }

    // To compute the CPU reference solution 

    hist_c = (unsigned int*)malloc(bsize);
    for(int i = 0; i < bins; i = i+1){
        hist_c[i] = 0;
    }

    // start the timer
    hipEventRecord(start,0);

    for(int i = 0; i < N; i = i+1) {
        index = (int)((data_h[i] - Rmin) / binsize);
        if( (index > bins - 1) || (index < 0)) {
            printf("data[%d]=%f, index=%d\n",i,data_h[i],index);
            exit(0);
        } 
        hist_c[index]++;
    }

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",2*N/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/(gputime_tot));

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check histogram sum equal to the total number of data 

    int sum = 0;
    for(int i = 0; i < bins; i = i+1) {
        sum += hist_c[i];
    }
    if(sum != N) {
        printf("Error, sum = %d\n",sum);
        exit(0);
    }

    // compare histograms from CPU and GPU

    for (int i = 0; i < bins; i++) {
        if(hist_h[i] != hist_c[i]) {
            printf("i=%d, hist_h=%d, hist_c=%d \n", i, hist_h[i], hist_c[i]);
        }
    }

    // save histogram in file
    FILE *out1;            
    out1 = fopen("hist_cpu.dat","w");

    fprintf(out1, "Histogram (CPU):\n");
    for(int i=0; i<bins; i++) {
        float x=Rmin+(i+0.5)*binsize;         // the center of each bin
        fprintf(out1,"%f %d \n",x,hist_c[i]);
    }
    fclose(out1);

    printf("Histogram (CPU):\n");
    for(int i=0; i<bins; i++) {
      float x=Rmin+(i+0.5)*binsize;         // the center of each bin
      printf("%f %d \n",x,hist_c[i]);
    }

    hipDeviceReset();

    free(data_h);
    free(hist_h);
    free(hist_c);

    return 0;
}


void RandomUniform(float* data, long n)   // RNG with uniform distribution in (0,1)
{
    for(long i = 0; i < n; i++){
        data[i] = rand()/(float)RAND_MAX;
    }
}

void RandomNormal(float* data, long n)   // RNG with normal distribution, mu=0, sigma=1
{
    const float Pi = acos(-1.0);

    for(long i = 0; i < n; i++) {
        double y = (double) rand() / (float)RAND_MAX;
        double x = -log(1.0-y);
        double z = (double) rand() / (float)RAND_MAX;
        double theta = 2*Pi*z;
        data[i] = (float) (sqrt(2.0*x)*cos(theta));   
    }
}


void RandomExpDecay(float* data, long n)   // RNG with Exponential Decay
{
    for(long i = 0; i < n; i = i+1){
        double y = (double) rand() / (float) RAND_MAX;
        data[i] = (float) -log(1.0 - y);
    }
}
