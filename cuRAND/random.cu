#include "hip/hip_runtime.h"
#include<unistd.h>
#include<stdio.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

__global__ void random(unsigned int seed, float* result, int N)
{
	hiprandState_t state;
	hiprand_init(seed, 0, 0, &state);

	*result = hiprand_uniform(&state);
	float ingpu_result;
	for(int i = 0; i < N; i = i+1){
		ingpu_result = hiprand_uniform(&state);
		printf("%lf\n", ingpu_result);	// the output series of this is not the same
		//printf("%lf\n", powf(ingpu_result, 2) );
	}
}

int main(){
	int gid;
	scanf("%d", &gid);
	hipSetDevice(gid);
	
	float x;
	float* gpu_x;
	int N = 10;
	hipMalloc((void**) &gpu_x, sizeof(float));
	
//	for (int i = 0; i < N; i = i+1){
//		random<<<1,1>>>(time(NULL), gpu_x, N);
//		hipMemcpy(&x, gpu_x, sizeof(float), hipMemcpyDeviceToHost);
	
//		printf("%lf\n", x);	// the output of this is the same

//	}

	random<<<1,1>>>(time(NULL), gpu_x, N);



	hipFree(gpu_x);

	hipDeviceReset();
	return 0;
}
